#include "hip/hip_runtime.h"
/****************************************************************************
* cuDFNsys - simulating flow and transport in 3D fracture networks          *
* Copyright (C) 2022, Tingchang YIN, Sergio GALINDO-TORRES                  *
*                                                                           *
* This program is free software: you can redistribute it and/or modify      *
* it under the terms of the GNU Affero General Public License as            *
* published by the Free Software Foundation, either version 3 of the        *
* License, or (at your option) any later version.                           *
*                                                                           *
* This program is distributed in the hope that it will be useful,           *
* but WITHOUT ANY WARRANTY; without even the implied warranty of            *
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the             *
* GNU Affero General Public License for more details.                       *
*                                                                           *
* You should have received a copy of the GNU Affero General Public License  *
* along with this program.  If not, see <https://www.gnu.org/licenses/>.    *
*****************************************************************************/

// ====================================================
// NAME:        DispersionAtOneDensityValue.cu
// DESCRIPTION: Dispersion in a DFN with a specific percolation parameter value
// AUTHOR:      Tingchang YIN
// DATE:        24/03/2023
// ====================================================

#include "cuDFNsys.cuh"
#include <fstream>
#include <iostream>
#include <unistd.h>

#ifdef USE_DOUBLES
typedef double _DataType_;
#else
typedef float _DataType_;
#endif

int main(int argc, char *argv[])
{
    // int A[3] ;
    // A[21546] = A[1];
    // return 0;

    double iStart = cuDFNsys::CPUSecond();

    // std::remove("./SimulationFailed.txt");
    // std::remove("./NoPercolation.txt");
    // std::remove("./SimulationFinished.txt");

    bool If_percolation_happens = false;
    try
    {
        int dev = 0;
        GPUErrCheck(hipSetDevice(dev));
        cuDFNsys::Warmup<<<256 / 256 + 1, 256 /*  1, 2*/>>>();
        hipDeviceSynchronize();

        time_t t;
        time(&t);

        srand((unsigned int)time(0));

        Eigen::MatrixXd Ter = Eigen::MatrixXd::Random(1, 1);

        int DSIZE = atoi(argv[1]);
        _DataType_ L = atof(argv[2]);
        _DataType_ kappa_ = atof(argv[3]),
                   beta_ = atof(argv[4]),
                   gamma_ = atof(argv[5]);
        int size_frac_mode = atoi(argv[6]); // mode of fracture size distributions
        cuDFNsys::Vector4<_DataType_> ParaSizeDistri =
            cuDFNsys::MakeVector4((_DataType_)atof(argv[7]),
                                  (_DataType_)atof(argv[8]),
                                  (_DataType_)atof(argv[9]),
                                  (_DataType_)atof(argv[10]));
        double3 DomainDimensionRatio = make_double3(1, 1, atof(argv[11]));
        int IfRemoveDeadEnd = atoi(argv[12]);
        _DataType_ minGridSize = atof(argv[13]);
        _DataType_ maxGridSize = atof(argv[14]);

        int NumTimeSteps_Dispersion = atoi(argv[15]);
        int NumParticlesRandomWalk = atoi(argv[16]);
        _DataType_ DeltaT = 0;
        _DataType_ Factor_mean_time_in_grid = atof(argv[17]);
        // the mean time (a characteristic grid length over the mean velocity (m/s)) for a random walk to cross a characteristic grid length
        // but this mean time was reduced, i.e., dividing by a factor (> 1)
        // then the mean time is DeltaT
        _DataType_ DiffusionLocal = 0;
        _DataType_ LengthScale_Over_Pe = 0;
        _DataType_ LengthScale = atof(argv[18]);
        _DataType_ Pe = atof(argv[19]);
        _DataType_ ControlPlaneSpacing = atof(argv[20]);
        bool IfoutputMsd = atoi(argv[21]) == 0 ? false : true;
        bool IfoutputParticleInfoAllsteps = atoi(argv[22]) == 0 ? false : true;
        string recordMode = IfoutputParticleInfoAllsteps == false ? "FPTCurve" : "OutputAll";
        _DataType_ P_in = L, P_out = 0;

        cout << "Number of fractures: " << DSIZE << endl;
        cout << "L: " << L << endl;
        cout << "Kappa: " << kappa_ << endl;
        cout << "Beta: " << beta_ << endl;
        cout << "Gamma: " << gamma_ << endl;
        cout << "Mode of fracture size distributions: " << size_frac_mode << endl;
        cout << "Parameters of the size distribution: " << ParaSizeDistri.x << ", " << ParaSizeDistri.y << ", " << ParaSizeDistri.z << ", " << ParaSizeDistri.w << endl;
        cout << "Domain's dimension ratio: " << DomainDimensionRatio.x << ", " << DomainDimensionRatio.y << ", " << DomainDimensionRatio.z << endl;
        cout << "If remove the dead ends: " << (IfRemoveDeadEnd == 0 ? "false" : "true") << endl;
        cout << "Min grid size: " << minGridSize << endl;
        cout << "Max grid size: " << maxGridSize << endl;
        cout << "Hydraulic head at the inlet and outlet: " << P_in << ", " << P_out << endl;
        cout << "Number of time steps for random walks: " << NumTimeSteps_Dispersion << endl;
        cout << "Number of particles: " << NumParticlesRandomWalk << endl;
        cout << "Factor_mean_time_in_grid: " << Factor_mean_time_in_grid << endl;
        cout << "LengthScale: " << LengthScale << endl;
        cout << "Pe: " << Pe << endl;
        cout << "The spacing of control planes: " << ControlPlaneSpacing << endl;
        cout << "IfoutputMsd: " << (IfoutputMsd == true ? "true" : "false") << endl;
        cout << "IfoutputParticleInfoAllsteps: " << (IfoutputParticleInfoAllsteps == false ? "FPTCurve" : "OutputAll") << endl;

        int perco_dir = 2;

        LengthScale_Over_Pe = LengthScale / Pe;
        string FractureFileName = "Fractures.h5";

        std::ifstream fileer(FractureFileName);
        bool pwqsc = fileer.good();

        if (!pwqsc) // no DFN is existing
        {
            thrust::host_vector<cuDFNsys::Fracture<_DataType_>> Frac_verts_host(DSIZE);
            thrust::device_vector<cuDFNsys::Fracture<_DataType_>> Frac_verts_device(DSIZE);
            cuDFNsys::Fracture<_DataType_> *Frac_verts_device_ptr;
            Frac_verts_device_ptr = thrust::raw_pointer_cast(Frac_verts_device.data());

            cuDFNsys::Fractures<_DataType_><<<DSIZE / 256 + 1, 256>>>(Frac_verts_device_ptr,
                                                                      (unsigned long)t + (unsigned long)ceil(abs(Ter(0, 0)) * ((unsigned long)t * 1.0)),
                                                                      DSIZE, L,
                                                                      0,
                                                                      ParaSizeDistri,
                                                                      kappa_, // kappa
                                                                      beta_,  // beta
                                                                      gamma_, // gamma
                                                                      DomainDimensionRatio);
            hipDeviceSynchronize();

            Frac_verts_host = Frac_verts_device;
            cout << "identifying intersections with complete fractures" << endl;
            std::map<pair<size_t, size_t>, pair<cuDFNsys::Vector3<_DataType_>, cuDFNsys::Vector3<_DataType_>>> Intersection_map;
            cuDFNsys::IdentifyIntersection<_DataType_> identifyInters{Frac_verts_host.size(),
                                                                      Frac_verts_device_ptr,
                                                                      false,
                                                                      Intersection_map};
            cout << "identifying cluster with complete fractures" << endl;
            std::vector<std::vector<size_t>> ListClusters;
            std::vector<size_t> Percolation_cluster;
            cuDFNsys::Graph<_DataType_> G{(size_t)DSIZE, Intersection_map};
            G.UseDFS(ListClusters);
            cuDFNsys::IdentifyPercolationCluster<_DataType_> IdentiClu{ListClusters,
                                                                       Frac_verts_host, perco_dir,
                                                                       Percolation_cluster};
            cout << "DFN I finished" << endl;
            cuDFNsys::MatlabPlotDFN<_DataType_> As{"DFN_I.h5", "DFN_I.m",
                                                   Frac_verts_host, Intersection_map, ListClusters,
                                                   Percolation_cluster, false, true, true, true,
                                                   L, perco_dir, true, "DFN_I", DomainDimensionRatio};
            cuDFNsys::OutputObjectData<_DataType_> lk;

            Intersection_map.clear();
            ListClusters.clear();
            Percolation_cluster.clear();
            cout << "identifying intersections with truncated fractures" << endl;
            cuDFNsys::IdentifyIntersection<_DataType_> identifyInters2{Frac_verts_host.size(),
                                                                       Frac_verts_device_ptr, true,
                                                                       Intersection_map};
            cout << "identifying cluster with truncated fractures" << endl;
            cuDFNsys::Graph<_DataType_> G2{(size_t)DSIZE, Intersection_map};
            G2.UseDFS(ListClusters);
            cuDFNsys::IdentifyPercolationCluster<_DataType_> IdentiClu2{ListClusters,
                                                                        Frac_verts_host, perco_dir,
                                                                        Percolation_cluster};
            cout << "DFN II finished" << endl;
            cuDFNsys::MatlabPlotDFN<_DataType_> As2{"DFN_II.h5", "DFN_II.m",
                                                    Frac_verts_host, Intersection_map, ListClusters,
                                                    Percolation_cluster, true, true, true, true,
                                                    L, perco_dir, true, "DFN_II", DomainDimensionRatio};
            //return 0;
            Frac_verts_device.clear();
            Frac_verts_device.shrink_to_fit();

            if (Percolation_cluster.size() > 0)
            {
                lk.OutputFractures("Fractures.h5", Frac_verts_host, L, DomainDimensionRatio);

                If_percolation_happens = true;
                // double istart_1 = cuDFNsys::CPUSecond();
                std::vector<size_t> Fracs_percol;

                cuDFNsys::GetAllPercolatingFractures GetPer{Percolation_cluster,
                                                            ListClusters,
                                                            Fracs_percol};
                std::vector<pair<int, int>> IntersectionPair_percol;
                int NUMprior = Fracs_percol.size();

                bool ifRemoveDeadEnds = (IfRemoveDeadEnd == 0 ? false : true);
                cout << "ifRemoveDeadEnds: " << ifRemoveDeadEnds << endl;
                cuDFNsys::RemoveDeadEndFrac<_DataType_> RDEF{Fracs_percol,
                                                             IntersectionPair_percol,
                                                             (size_t)perco_dir,
                                                             Frac_verts_host,
                                                             Intersection_map,
                                                             ifRemoveDeadEnds};

                if (ifRemoveDeadEnds)
                    cout << "remove " << NUMprior - Frac_verts_host.size() << " fractures\n";

                cout << "meshing ..." << endl;

                cuDFNsys::Mesh<_DataType_> mesh{Frac_verts_host, IntersectionPair_percol,
                                                &Fracs_percol, minGridSize, maxGridSize, perco_dir, L,
                                                DomainDimensionRatio};
                lk.OutputMesh("mesh.h5", mesh, Fracs_percol);

                double mean_grid_area = mesh.MatlabPlot("DFN_mesh_.h5",
                                                        "DFN_mesh_.m",
                                                        Frac_verts_host,
                                                        L, true, true, true,
                                                        "DFN_mesh_", DomainDimensionRatio);
                cout << "The mean area of all elements is " << mean_grid_area << endl;

                cout << "MHFEM ing ..." << endl;

                cuDFNsys::MHFEM<_DataType_> fem{mesh, Frac_verts_host, P_in, P_out,
                                                perco_dir, L, DomainDimensionRatio};
                lk.OutputMHFEM("mhfem.h5", fem);

                cout << "Fluxes: " << fem.QIn << ", ";
                cout << fem.QOut << ", Permeability: ";
                cout << fem.Permeability << endl;
                cout << "Error between the inlet and outlet fluxes: " << abs(fem.QIn - fem.QOut) / ((fem.QOut + fem.QIn) * 0.5) * 100.0 << "%\n";
                if (fem.QError > 1 || isnan(fem.Permeability) == 1)
                    throw cuDFNsys::ExceptionsIgnore("Found large error or isnan, the error: " + std::to_string(fem.QError) + ", the permeability: " + std::to_string(fem.Permeability) + "\n");

                //---------------------
                double2 TGH = fem.MatlabPlot("MHFEM_.h5",
                                             "MHFEM_.m",
                                             Frac_verts_host, mesh, L, true, "MHFEM_", DomainDimensionRatio);

                double meanV = TGH.x;
                double maxV = TGH.y;

                cout << "The maximum velocity of all elements is " << maxV << endl;
                cout << "The mean velocity of all elements is " << meanV << endl;

                double meanTime = pow(mean_grid_area, 0.5) / maxV;

                DeltaT = meanTime / Factor_mean_time_in_grid;

                cout << "\nThe delta T is set to be " << ("\033[1;33m") << DeltaT << ("\033[0m") << "\n\n";

                DiffusionLocal = LengthScale_Over_Pe * meanV;
                cout << "\nThe DiffusionLocal is set to be " << ("\033[1;33m") << DiffusionLocal << ("\033[0m") << "\n\n";
                //---------------

                string Filename_FracturesForParticle = "FracturesForParticle.h5";

                std::ifstream file(Filename_FracturesForParticle);
                bool pwqs = file.good();

                if (!pwqs)
                {
                    cout << "Writting " << Filename_FracturesForParticle << endl;
                    cuDFNsys::OutputObjectData<_DataType_> lk;
                    lk.OutputFractures(Filename_FracturesForParticle, Frac_verts_host, L, DomainDimensionRatio);
                }

                cout << "Particle transport ing ......\n";

                cuDFNsys::ParticleTransport<_DataType_> p{NumTimeSteps_Dispersion, // number of time step
                                                          Frac_verts_host,
                                                          mesh,
                                                          fem,
                                                          (uint)perco_dir,
                                                          -0.5f * L * (&DomainDimensionRatio.x)[perco_dir],
                                                          NumParticlesRandomWalk, // num of particle
                                                          DeltaT,                 // delta_T_ii
                                                          DiffusionLocal,
                                                          "Particle_tracking",
                                                          "Flux-weighted",
                                                          recordMode,
                                                          false, 1, false,
                                                          ControlPlaneSpacing, IfoutputMsd};

                p.MatlabPlot("MHFEM_.h5", "ParticlesDFNMatlab.m", mesh, fem, L, DomainDimensionRatio, true, "ParticlesDFN");
            }
        }
        else // if the file exists
        {
            cout << "preparing" << endl;

            thrust::host_vector<cuDFNsys::Fracture<_DataType_>> Frac_verts_host;
            thrust::device_vector<cuDFNsys::Fracture<_DataType_>> Frac_verts_device;

            cuDFNsys::InputObjectData<_DataType_> lk;
            lk.InputFractures("Fractures.h5", Frac_verts_host, L, DomainDimensionRatio);

            DSIZE = Frac_verts_host.size();

            Frac_verts_device = Frac_verts_host;
            cuDFNsys::Fracture<_DataType_> *Frac_verts_device_ptr;
            Frac_verts_device_ptr = thrust::raw_pointer_cast(Frac_verts_device.data());

            cout << "identifying intersections with complete fractures" << endl;
            std::map<pair<size_t, size_t>, pair<cuDFNsys::Vector3<_DataType_>, cuDFNsys::Vector3<_DataType_>>> Intersection_map;
            cuDFNsys::IdentifyIntersection<_DataType_> identifyInters{Frac_verts_host.size(),
                                                                      Frac_verts_device_ptr,
                                                                      false,
                                                                      Intersection_map};
            cout << "identifying cluster with complete fractures" << endl;
            std::vector<std::vector<size_t>> ListClusters;
            std::vector<size_t> Percolation_cluster;
            cuDFNsys::Graph<_DataType_> G{(size_t)DSIZE, Intersection_map};
            G.UseDFS(ListClusters);
            cuDFNsys::IdentifyPercolationCluster<_DataType_> IdentiClu{ListClusters,
                                                                       Frac_verts_host, perco_dir,
                                                                       Percolation_cluster};
            cout << "DFN I finished" << endl;
            cuDFNsys::MatlabPlotDFN<_DataType_> As{"DFN_I.h5", "DFN_I.m",
                                                   Frac_verts_host, Intersection_map, ListClusters,
                                                   Percolation_cluster, false, true, true, true,
                                                   L, perco_dir, true, "DFN_I", DomainDimensionRatio};

            //
            Intersection_map.clear();
            ListClusters.clear();
            Percolation_cluster.clear();
            cout << "identifying intersections with truncated fractures" << endl;
            cuDFNsys::IdentifyIntersection<_DataType_> identifyInters2{Frac_verts_host.size(),
                                                                       Frac_verts_device_ptr, true,
                                                                       Intersection_map};
            cout << "identifying cluster with truncated fractures" << endl;
            cuDFNsys::Graph<_DataType_> G2{(size_t)DSIZE, Intersection_map};
            G2.UseDFS(ListClusters);
            cuDFNsys::IdentifyPercolationCluster<_DataType_> IdentiClu2{ListClusters,
                                                                        Frac_verts_host, perco_dir,
                                                                        Percolation_cluster};
            cout << "DFN II finished" << endl;
            cuDFNsys::MatlabPlotDFN<_DataType_> As2{"DFN_II.h5", "DFN_II.m",
                                                    Frac_verts_host, Intersection_map, ListClusters,
                                                    Percolation_cluster, true, true, true, true,
                                                    L, perco_dir, true, "DFN_II", DomainDimensionRatio};
            Frac_verts_device.clear();
            Frac_verts_device.shrink_to_fit();
            //-----------
            if (Percolation_cluster.size() > 0)
            {
                If_percolation_happens = true;
                double istart_1 = cuDFNsys::CPUSecond();
                std::vector<size_t> Fracs_percol;
                cuDFNsys::GetAllPercolatingFractures GetPer{Percolation_cluster,
                                                            ListClusters,
                                                            Fracs_percol};
                std::vector<pair<int, int>> IntersectionPair_percol;

                bool ifRemoveDeadends = (IfRemoveDeadEnd == 0 ? false : true);

                cuDFNsys::RemoveDeadEndFrac<_DataType_> RDEF{Fracs_percol,
                                                             IntersectionPair_percol,
                                                             (size_t)perco_dir,
                                                             Frac_verts_host,
                                                             Intersection_map, ifRemoveDeadends};
                cout << "meshing ..." << endl;

                cuDFNsys::OutputObjectData<_DataType_> lkew;
                lkew.OutputFractures("FracturesII.h5", Frac_verts_host, L, DomainDimensionRatio);

                cuDFNsys::Mesh<_DataType_> mesh;
                try
                {
                    lk.InputMesh("mesh.h5", mesh, &Fracs_percol);
                }
                catch (...)
                {
                    cout << "mesh ing ...\n";
                    cuDFNsys::Mesh<_DataType_> mesh2{Frac_verts_host, IntersectionPair_percol,
                                                     &Fracs_percol, minGridSize,
                                                     maxGridSize, perco_dir, L, DomainDimensionRatio};
                    lkew.OutputMesh("mesh.h5", mesh2, Fracs_percol);
                    lk.InputMesh("mesh.h5", mesh, &Fracs_percol);
                }

                double mean_grid_area = mesh.MatlabPlot("DFN_mesh_.h5",
                                                        "DFN_mesh_.m",
                                                        Frac_verts_host,
                                                        L, true, true, true,
                                                        "DFN_mesh_", DomainDimensionRatio);
                cout << "The mean area of all elements is " << mean_grid_area << endl;
                cout << "MHFEM ing ..." << endl;

                cuDFNsys::MHFEM<_DataType_> fem;
                try
                {
                    cout << "Loading mhfem ...\n";
                    lk.InputMHFEM("mhfem.h5", fem);
                }
                catch (...)
                {
                    cuDFNsys::MHFEM<_DataType_> fem2{mesh, Frac_verts_host, P_in, P_out, perco_dir, L, DomainDimensionRatio};

                    lkew.OutputMHFEM("mhfem.h5", fem2);
                    fem = fem2;
                };

                cout << "Fluxes: " << fem.QIn << ", ";
                cout << fem.QOut << ", Permeability: ";
                cout << fem.Permeability << endl;
                if (fem.QError > 1 || isnan(fem.Permeability) == 1)
                    throw cuDFNsys::ExceptionsIgnore("Found large error or isnan, the error: " + std::to_string(fem.QError) + ", the permeability: " + std::to_string(fem.Permeability) + "\n");

                //---------------------
                double2 TGH = fem.MatlabPlot("MHFEM_.h5",
                                             "MHFEM_.m",
                                             Frac_verts_host, mesh, L, true, "MHFEM_", DomainDimensionRatio);

                double meanV = TGH.x;
                double maxV = TGH.y;

                cout << "The maximum velocity of all elements is " << maxV << endl;
                cout << "The mean velocity of all elements is " << meanV << endl;

                double meanTime = pow(mean_grid_area, 0.5) / maxV;

                DeltaT = meanTime / Factor_mean_time_in_grid;

                cout << "\nThe delta T is set to be " << ("\033[1;33m") << DeltaT << ("\033[0m") << "\n\n";

                DiffusionLocal = LengthScale_Over_Pe * meanV;
                cout << "\nThe DiffusionLocal is set to be " << ("\033[1;33m") << DiffusionLocal << ("\033[0m") << "\n\n";
                //-----------------

                string FractureFileName_r = "ParticlePositionResult/DispersionInfo.h5";

                std::ifstream fileeqr(FractureFileName_r);
                bool psd = fileeqr.good();

                if (psd)
                {
                    cuDFNsys::HDF5API hg6;
                    vector<double> Aqs = hg6.ReadDataset<double>(FractureFileName_r,
                                                                 "N", "Delta_T");
                    cout << "\nThe delta T is set to be " << ("\033[1;33m") << Aqs[0] << ("\033[0m") << "\n\n";

                    Aqs = hg6.ReadDataset<double>(FractureFileName_r,
                                                  "N", "Dispersion_local");
                    cout << "\nThe DiffusionLocal is set to be " << ("\033[1;33m") << Aqs[0] << ("\033[0m") << "\n\n";
                }
                //---------------
                // return 0;

                string Filename_FracturesForParticle = "FracturesForParticle.h5";

                std::ifstream file(Filename_FracturesForParticle);
                bool pwqs = file.good();

                if (!pwqs)
                {
                    cout << "Writting " << Filename_FracturesForParticle << endl;
                    cuDFNsys::OutputObjectData<_DataType_> lk;
                    lk.OutputFractures(Filename_FracturesForParticle, Frac_verts_host, L);
                }
                cout << "Particle transport ing ......\n";

                cuDFNsys::ParticleTransport<_DataType_> p{NumTimeSteps_Dispersion, // number of time step
                                                          Frac_verts_host, mesh, fem, (uint)perco_dir,
                                                          -0.5f * L * (&DomainDimensionRatio.x)[perco_dir],
                                                          NumParticlesRandomWalk, // num of particle
                                                          DeltaT,                 // delta_T_ii
                                                          DiffusionLocal,
                                                          "Particle_tracking",
                                                          "Flux-weighted",
                                                          recordMode,
                                                          false, 1, false, ControlPlaneSpacing, IfoutputMsd};
                p.MatlabPlot("MHFEM_.h5", "ParticlesDFNMatlab.m", mesh, fem, L, DomainDimensionRatio, true, "ParticlesDFN");
            }
            //hipDeviceReset();
        }
    }
    catch (cuDFNsys::ExceptionsIgnore &e)
    {
        cout << e.what() << endl;
        cout << "Failed simulation!\n";
        // std::ofstream fs("./SimulationFailed.txt");
        // fs.close();
        exit(0);
    }
    catch (cuDFNsys::ExceptionsPause &e)
    {
        cout << e.what() << endl;
        cout << "Failed simulation!\n";
        // std::ofstream fs("./SimulationFailed.txt");
        // fs.close();
        exit(0);
    }
    catch (...)
    {
        cout << "Failed simulation!\n";
        // std::ofstream fs("./SimulationFailed.txt");
        // fs.close();
        exit(0);
    }

    if (!If_percolation_happens)
    {
        cout << "No percolation happens\n";
        // std::ofstream fs("./NoPercolation.txt");
        // fs.close();
    }

    // std::ofstream fs("./SimulationFinished.txt");
    // fs.close();

    cout << "This simulation consumes " << cuDFNsys::CPUSecond() - iStart << " seconds\n";
    return 0;
};
