/****************************************************************************
* cuDFNsys - simulating flow and transport in 3D fracture networks          *
* Copyright (C) 2022, Tingchang YIN, Sergio GALINDO-TORRES                  *
*                                                                           *
* This program is free software: you can redistribute it and/or modify      *
* it under the terms of the GNU Affero General Public License as            *
* published by the Free Software Foundation, either version 3 of the        *
* License, or (at your option) any later version.                           *
*                                                                           *
* This program is distributed in the hope that it will be useful,           *
* but WITHOUT ANY WARRANTY; without even the implied warranty of            *
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the             *
* GNU Affero General Public License for more details.                       *
*                                                                           *
* You should have received a copy of the GNU Affero General Public License  *
* along with this program.  If not, see <https://www.gnu.org/licenses/>.    *
*****************************************************************************/

// ====================================================
// NAME:        A test case
// DESCRIPTION: Call cuDFNsys functions to do simulation and test.
// AUTHOR:      Tingchang YIN
// DATE:        30/06/2022
// ====================================================

#include "cuDFNsys.cuh"
#include <unistd.h>

int main(int argc, char *argv[])
{
    try
    {
        string nameDFN = string(argv[1]);

        cuDFNsys::DFN<double> myDFN;
        myDFN.LoadClassFromH5(nameDFN);

        myDFN.Visualization("DFNVisual", "DFNVisual", "DFNVisual", false, true, true, true);
    }
    catch (cuDFNsys::ExceptionsIgnore &e)
    {
        cout << e.what() << endl;
    }
    catch (cuDFNsys::ExceptionsPause &e)
    {
        cout << e.what() << endl;
    }
    catch (...)
    {
        throw;
    };
    return 0;
};