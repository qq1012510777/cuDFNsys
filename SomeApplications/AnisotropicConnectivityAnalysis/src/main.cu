#include "hip/hip_runtime.h"
#include "cuDFNsys.cuh"
#include <algorithm>
#include <chrono>
#include <cstdlib> // For std::system
#include <ctime>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <limits.h>
#include <memory>
#include <numeric>
#include <random>
#include <sstream>
#include <string>
#include <thread>
#include <unistd.h>
#include <vector>
using namespace std;
namespace fs = std::filesystem;

bool RunCMD_with_RealTimeCheck(const string &cmd, const string &logFile,
                               const bool &IfGenLogFile = false);
void CreateOrEmptyFile(const std::string &filename);
void AddLineToFile(const std::string &filename, const std::string &line);
bool IfAFileExist(const string &FilePath);
string DoubleNumberToScientificNotationString(const double &number);
uint GetH5DatasetSize(const string &nameH5, const string &nameDataset);

std::ostream &operator<<(std::ostream &os, const double3 &vec)
{
    os << vec.x << ", " << vec.y << ", " << vec.z;
    return os;
}
std::ostream &operator<<(std::ostream &os, const double4 &vec)
{
    os << vec.x << ", " << vec.y << ", " << vec.z << ", " << vec.w;
    return os;
}

bool ErrorCheck(int &erroCounter, const int &errorCountLimit)
{
    erroCounter++;
    if (erroCounter > errorCountLimit)
    {
        std::cout << "Too many errors happen\n";
        return true;
    }
    return false;
};

int main(int argc, char *argv[])
{
    time_t t;
    time(&t);

    // string ExeuctablePath = argv[1];
    double DomainSizeX = atof(argv[1]);
    double3 DomainDimensionRatio =
        make_double3(atof(argv[2]), atof(argv[3]), atof(argv[4]));

    int NumGroups = atoi(argv[5]);

    std::vector<double> Kappa_input(NumGroups, 0);
    std::vector<cuDFNsys::Vector3<double>> MeanOrientationOfFisherDistribution_input(NumGroups);
    std::vector<int> FracNumInit_input(NumGroups, 0),
        FracNumIncre_input(NumGroups, 0),
        NumFracIncre_input(NumGroups, 0);
    std::vector<int> ModeOfSizeDistribution_input(NumGroups, 0);
    std::vector<cuDFNsys::Vector4<double>> SizeDistributionParameters_input(NumGroups);
    std::vector<double> Beta_input(NumGroups, 0), Gamma_input(NumGroups, 0);

    for (int i = 0; i < NumGroups; ++i)
    {
        int index = 14 * i;

        Kappa_input[i] = atof(argv[6 + index]);
        MeanOrientationOfFisherDistribution_input[i] = make_double3(
            atof(argv[7 + index]), atof(argv[8 + index]), atof(argv[9 + index]));
        FracNumInit_input[i] = atoi(argv[10 + index]);
        FracNumIncre_input[i] = atoi(argv[11 + index]);
        NumFracIncre_input[i] = atoi(argv[12 + index]);

        ModeOfSizeDistribution_input[i] = atoi(argv[13 + index]);

        SizeDistributionParameters_input[i] = make_double4(
            atof(argv[14 + index]), atof(argv[15 + index]),
            atof(argv[16 + index]), atof(argv[17 + index]));

        Beta_input[i] = atof(argv[18 + index]);
        Gamma_input[i] = atof(argv[19 + index]);
    }
    std::cout << "Domain Size = " << DomainSizeX << endl;
    cout << "Domain dimension ratio: " << DomainDimensionRatio << endl;
    cout << "Number of fracture groups: " << NumGroups << endl;

    for (int i = 0; i < NumGroups; ++i)
    {
        cout << "---------group " << i + 1 << "----------------\n";
        cout << "Kappa: " << Kappa_input[i] << endl;
        cout << "Mean orientation: " << MeanOrientationOfFisherDistribution_input[i] << endl;
        cout << "Init fracture number: " << FracNumInit_input[i] << endl;
        cout << "Facture number increament: " << FracNumIncre_input[i] << endl;
        cout << "Number of increments: " << NumFracIncre_input[i] << endl;
        cout << "ModeOfSizeDistribution_input: " << ModeOfSizeDistribution_input[i] << endl;
        cout << "SizeDistributionParameters_input: " << SizeDistributionParameters_input[i] << endl;
        cout << "Beta_input: " << Beta_input[i] << endl;
        cout << "Gamma_input: " << Gamma_input[i] << endl;
        cout << "--------------------------------\n\n";
    }
    //--------------------------------------
    int errorCountLimit = 5;
    int erroCounter = 0;

    for (int i = 0; i <= NumFracIncre_input[0]; ++i)
    {
        try
        {
            cout << "\n\n---------------loop " << i + 1 << endl;
            /* code */
            cuDFNsys::DFN<double> my_dfn;
            my_dfn.RandomSeed = (unsigned long)t;
            my_dfn.DomainSizeX = DomainSizeX;
            my_dfn.DomainDimensionRatio = DomainDimensionRatio;

            my_dfn.NumFractures.resize(NumGroups);
            my_dfn.Kappa.resize(NumGroups);
            my_dfn.MeanOrientationOfFisherDistribution.resize(NumGroups);
            my_dfn.Beta.resize(NumGroups);
            my_dfn.Gamma.resize(NumGroups);
            my_dfn.ModeOfSizeDistribution.resize(NumGroups);
            my_dfn.SizeDistributionParameters.resize(NumGroups);

            for (int j = 0; j < NumGroups; ++j)
            {
                my_dfn.NumFractures[j] = FracNumInit_input[j] + i * FracNumIncre_input[j];
                my_dfn.Kappa[j] = Kappa_input[j];
                my_dfn.MeanOrientationOfFisherDistribution[j] = MeanOrientationOfFisherDistribution_input[j];
                my_dfn.Beta[j] = Beta_input[j];
                my_dfn.Gamma[j] = Gamma_input[j];
                my_dfn.ModeOfSizeDistribution[j] = ModeOfSizeDistribution_input[j];
                my_dfn.SizeDistributionParameters[j] = SizeDistributionParameters_input[j];
            }

            bool IfReRun = false;
            for (int percoDir = 0; percoDir < 3; ++percoDir)
            {
                if (IfAFileExist("ClassDFN_" + cuDFNsys::ToStringWithWidth(i, 2) + "_" +
                                 cuDFNsys::ToStringWithWidth(percoDir, 1) + ".h5"))
                {
                    cout << ("ClassDFN_" + cuDFNsys::ToStringWithWidth(i, 2) + "_" +
                             cuDFNsys::ToStringWithWidth(percoDir, 1) + ".h5")
                         << " exists\n";
                    continue;
                }
                IfReRun = true;
            }

            if (IfReRun)
            {
                cout << "This loop is running\n";
                for (int percoDir = 0; percoDir < 3; ++percoDir)
                {

                    cuDFNsys::DFN<double> my_dfn_2;
                    my_dfn_2 = my_dfn;

                    my_dfn_2.PercoDir = percoDir;
                    my_dfn_2.FractureGeneration();
                    my_dfn_2.IdentifyIntersectionsClusters(false);

                    my_dfn_2.StoreInH5("ClassDFN_" + cuDFNsys::ToStringWithWidth(i, 2) + "_" +
                                       cuDFNsys::ToStringWithWidth(percoDir, 1));
                }
            }

            erroCounter = 0;
        }
        catch (cuDFNsys::ExceptionsIgnore e)
        {
            cout << "cuDFNsys::Exceptions: " << e.what() << endl;
            int result_system = system(
                std::string("rm -rf ./ClassDFN_" + cuDFNsys::ToStringWithWidth(i, 2) + "_*.h5")
                    .c_str());

            i--;
            if (ErrorCheck(erroCounter, errorCountLimit))
                break;
            continue;
        }
        catch (cuDFNsys::ExceptionsPause e)
        {
            cout << "cuDFNsys::Exceptions: " << e.what() << endl;
            int result_system = system(
                std::string("rm -rf ./ClassDFN_" + cuDFNsys::ToStringWithWidth(i, 2) + "_*.h5")
                    .c_str());
            i--;
            if (ErrorCheck(erroCounter, errorCountLimit))
                break;
            continue;
        }
        catch (H5::Exception e)
        {
            cout << "H5::Exceptions: " << e.getDetailMsg() << endl;
            int result_system = system(
                std::string("rm -rf ./ClassDFN_" + cuDFNsys::ToStringWithWidth(i, 2) + "_*.h5")
                    .c_str());
            i--;
            if (ErrorCheck(erroCounter, errorCountLimit))
                break;
            continue;
        }
        catch (H5::FileIException e)
        {
            cout << "H5::Exceptions: " << e.getDetailMsg() << endl;
            int result_system = system(
                std::string("rm -rf ./ClassDFN_" + cuDFNsys::ToStringWithWidth(i, 2) + "_*.h5")
                    .c_str());
            i--;
            if (ErrorCheck(erroCounter, errorCountLimit))
                break;
            continue;
        }
        catch (...)
        {
            cout << "Unknown exceptions\n";
            int result_system = system(
                std::string("rm -rf ./ClassDFN_" + cuDFNsys::ToStringWithWidth(i, 2) + "_*.h5")
                    .c_str());
            i--;
            if (ErrorCheck(erroCounter, errorCountLimit))
                break;
            continue;
        }
    }
    return 0;
}

bool RunCMD_with_RealTimeCheck(const string &cmd, const string &logFile,
                               const bool &IfGenLogFile)
{
    if (IfGenLogFile)
        CreateOrEmptyFile(logFile);

    FILE *pipe = popen((cmd + " 2>&1").c_str(), "r");
    if (!pipe)
    {
        // std::cerr << "popen() failed!";
        cout << "command failed: " << cmd << endl;
        return false;
    }

    // Read the output line by line
    char buffer[1024];
    bool statu = true;

    while (fgets(buffer, 1024, pipe) != nullptr)
    {
        std::string output(buffer);
        // std::cout << output; // Print the output in real-time
        AddLineToFile(logFile, output);
        // Check if the output contains any warning
        if (output.find("Warning") != std::string::npos)
        {
            // std::cerr << "Warning detected!\n";
            //  Take appropriate action
            pclose(pipe);
            statu = false;
            return statu;
        }
    }

    // Close the pipe
    pclose(pipe);
    return statu;
}

void CreateOrEmptyFile(const std::string &filename)
{
    std::ofstream file(filename, std::ios::out | std::ios::trunc);
    if (file.is_open())
    {
        // std::cout << "File created or emptied successfully: " << filename
        //          << std::endl;
        file.close();
    }
    else
    {
        std::cerr << "Failed to create or empty file: " << filename
                  << std::endl;
    }
}
void AddLineToFile(const std::string &filename, const std::string &line)
{
    std::ofstream file(filename, std::ios::out | std::ios::app);
    if (file.is_open())
    {
        file << line;
        // std::cout << "Line added to file: " << filename << std::endl;
        file.close();
    }
    else
    {
        std::cerr << "Failed to open file: " << filename << std::endl;
    }
}
bool IfAFileExist(const string &FilePath)
{
    fs::path filePath = FilePath;

    // Check if the file exists
    if (fs::exists(filePath))
    {
        return true;
    }
    else
    {
        return false;
    }
}
string DoubleNumberToScientificNotationString(const double &number)
{
    std::stringstream ss;
    ss << std::scientific << number;
    std::string result = ss.str();
    return result;
}

uint GetH5DatasetSize(const string &nameH5, const string &nameDataset)
{
    H5File file(nameH5, H5F_ACC_RDONLY);

    // Open the dataset
    DataSet dataset = file.openDataSet(nameDataset);

    // Get the dataspace of the dataset
    DataSpace dataspace = dataset.getSpace();

    // Get the number of dimensions in the dataspace
    int ndims = dataspace.getSimpleExtentNdims();

    // Create a vector to store the size of each dimension
    hsize_t dims[ndims];

    // Get the size of each dimension
    dataspace.getSimpleExtentDims(dims);

    // Calculate the total size of the dataset
    uint totalSize = 1;
    for (int i = 0; i < ndims; ++i)
    {
        totalSize *= dims[i];
    }

    // Output the total size
    // std::cout << "Size of dataset: " << totalSize << std::endl;

    // Close the dataset and file
    dataset.close();
    file.close();

    return totalSize;
}
