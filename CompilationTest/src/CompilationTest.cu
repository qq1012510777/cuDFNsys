#include "cuDFNsys.cuh"
int main()
{
    int dev = 0;
    GPUErrCheck(hipSetDevice(dev));
    cuDFNsys::Vector4<double> vec4 = cuDFNsys::MakeVector4(1.5, 1., 15., 0.);
    std::cout << "cuDFNsys::Vector4: " << vec4.x << ", " << vec4.y << ", " << vec4.z << ", " << vec4.w << ", " << std::endl;
    return 0;
}