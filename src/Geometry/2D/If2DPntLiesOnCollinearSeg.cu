#include "hip/hip_runtime.h"
#include "Geometry/2D/If2DPntLiesOnCollinearSeg.cuh"

// ====================================================
// NAME:        If2DPntLiesOnCollinearSeg
// DESCRIPTION: check if a 2D point lies on a collinear segment
//              point q, line segment: p-r
// AUTHOR:      Tingchang YIN
// DATE:        07/05/2022
// ====================================================
template <typename T>
__device__ __host__ bool cuDFNsys::If2DPntLiesOnCollinearSeg(cuDFNsys::Vector2<T> p, cuDFNsys::Vector2<T> q, cuDFNsys::Vector2<T> r)
{
    if (q.x <= max(p.x, r.x) && q.x >= min(p.x, r.x) &&
        q.y <= max(p.y, r.y) && q.y >= min(p.y, r.y))
        return true;

    return false;
}; // If2DPntLiesOnCollinearSeg
template __device__ __host__ bool cuDFNsys::If2DPntLiesOnCollinearSeg<double>(cuDFNsys::Vector2<double> p, cuDFNsys::Vector2<double> q, cuDFNsys::Vector2<double> r);
template __device__ __host__ bool cuDFNsys::If2DPntLiesOnCollinearSeg<float>(cuDFNsys::Vector2<float> p, cuDFNsys::Vector2<float> q, cuDFNsys::Vector2<float> r);