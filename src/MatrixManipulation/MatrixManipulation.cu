#include "hip/hip_runtime.h"
#include "MatrixManipulation/MatrixManipulation.cuh"

// ====================================================
// NAME:        CrossProductFloat3
// DESCRIPTION: Cross product of two 3D vectors.
// AUTHOR:      Tingchang YIN
// DATE:        04/04/2022
// ====================================================
__device__ __host__ float3 cuDFNsys::CrossProductFloat3(float3 v1, float3 v2)
{
    float3 n = make_float3(v1.y * v2.z - v1.z * v2.y,
                           v1.z * v2.x - v1.x * v2.z,
                           v1.x * v2.y - v1.y * v2.x);
    return n;
}; // CrossProductFloat3

// ====================================================
// NAME:        CrossProductFloat2
// DESCRIPTION: because the A and B are in XY plane,
//              so the cross product is
//              (0, 0, A.x * B.y - B.x * A.y).
// AUTHOR:      Tingchang YIN
// DATE:        08/04/2022
// ====================================================
__device__ __host__ float cuDFNsys::CrossProductFloat2(float2 A, float2 B)
{
    return A.x * B.y - B.x * A.y;
}

// ====================================================
// NAME:        ProductSquare3Float3
// DESCRIPTION: Product of a square matrix and a column vector.
// AUTHOR:      Tingchang YIN
// DATE:        07/04/2022
// ====================================================
__device__ __host__ float3 cuDFNsys::ProductSquare3Float3(float A[3][3], float3 B)
{
    float B_[3] = {B.x, B.y, B.z};
    float C[3] = {0};

    for (int i = 0; i < 3; ++i)
    {
        for (int j = 0; j < 3; ++j)
        {
            C[i] += A[i][j] * B_[j];
        }
    }

    float3 C_ = make_float3(C[0], C[1], C[2]);

    return C_;
}; // ProductSquare3Float3

// ====================================================
// NAME:        ProjectVToPlaneN
// DESCRIPTION: project a vector V to a plane which has normal of n
// AUTHOR:      Tingchang YIN
// DATE:        12/05/2022
// ====================================================
__device__ __host__ float3 cuDFNsys::ProjectVToPlaneN(float3 V, float3 n)
{
    float I_minus_nn[3][3] = {1.0f - n.x * n.x, -n.x * n.y, -n.x * n.z,
                              -n.y * n.x, 1.0f - n.y * n.y, -n.y * n.z,
                              -n.z * n.x, -n.z * n.y, 1.0f - n.z * n.z};

    float3 KL = cuDFNsys::ProductSquare3Float3(I_minus_nn, V);
    float norm_KL = sqrt(KL.x * KL.x + KL.y * KL.y + KL.z * KL.z);

    KL.x /= norm_KL;
    KL.y /= norm_KL;
    KL.z /= norm_KL;

    return KL;
}; // ProjectVToPlaneN