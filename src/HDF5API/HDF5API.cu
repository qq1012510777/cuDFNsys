#include "hip/hip_runtime.h"
#include "HDF5API/HDF5API.cuh"

// ====================================================
// NAME:        NewFile
// DESCRIPTION: create a new h5 file
// AUTHOR:      Tingchang YIN
// DATE:        09/04/2022
// ====================================================
void cuDFNsys::HDF5API::NewFile(const string &name)
{
    try
    {
        H5::Exception::dontPrint();
        H5File file(name, H5F_ACC_TRUNC);
        file.close();
    }
    catch (...)
    {
        cout << "\033[31mA file with same name does exist and is occupied by anthor program now!\033[0m\n";
        cout << "\033[31mSo, I will delete this file and create a new one!\033[0m\n";
        string name1 = name;
        std::remove(name1.c_str());
        H5File file(name, H5F_ACC_TRUNC);
        file.close();
    }
}; // NewFile

// ====================================================
// NAME:        AddDataset
// DESCRIPTION: add dataset to a h5 file
// AUTHOR:      Tingchang YIN
// DATE:        09/04/2022
// ====================================================
template <class T>
void cuDFNsys::HDF5API::AddDataset(const string &filename,
                                   const string &groupname,
                                   const string &datasetname,
                                   const T *data, // the T should be column-major
                                   const uint2 &dim)
{
    H5File file(filename, H5F_ACC_RDWR);

    hsize_t *dims; // dataset dimensions for each rank
    int rank_ = 1;
    if (dim.y > 1)
        rank_ = 2;

    dims = new hsize_t[rank_];
    dims[0] = dim.x;
    if (rank_ == 2)
        dims[1] = dim.y;
    DataSpace dataspace(rank_, dims);
    delete[] dims;
    dims = NULL;

    if (groupname != "N")
    {
        Group group;

        H5::Exception::dontPrint();
        try
        {
            //cout << "try to open a group!\n";
            group = file.openGroup(groupname);
            //cout << "opened group!\n";
        }
        catch (...)
        {
            //cout << "no this group! create a new group!\n";
            group = file.createGroup(groupname);
            //cout << "created group!\n";
        }

        DataSet dataset =
            group.createDataSet(datasetname, PredType::NATIVE_DOUBLE, dataspace);

        double *buffer = new double[dim.x * dim.y]();
        if (buffer == NULL)
        {
            string AS = "Alloc error in HDF5API::AddDataset!\n";
            group.close();
            file.close();
            throw ExceptionsPause(AS);
        }

        for (size_t i = 0; i < dim.x * dim.y; ++i)
            buffer[i] = data[i];

        dataset.write(buffer, PredType::NATIVE_DOUBLE);

        delete[] buffer;
        buffer = NULL;

        group.close();
    }
    else
    {
        DataSet dataset =
            file.createDataSet(datasetname, PredType::NATIVE_DOUBLE, dataspace);

        double *buffer = new double[dim.x * dim.y]();
        if (buffer == NULL)
        {
            string AS = "Alloc error in HDF5API::AddDataset!\n";
            file.close();
            throw ExceptionsPause(AS);
        }

        for (size_t i = 0; i < dim.x * dim.y; ++i)
            buffer[i] = data[i];

        dataset.write(buffer, PredType::NATIVE_DOUBLE);

        delete[] buffer;
        buffer = NULL;
    }

    file.close();
}; // AddDataset
template void cuDFNsys::HDF5API::AddDataset(const string &filename, const string &groupname, const string &datasetname, const int *data, const uint2 &dim);
template void cuDFNsys::HDF5API::AddDataset(const string &filename, const string &groupname, const string &datasetname, const double *data, const uint2 &dim);
template void cuDFNsys::HDF5API::AddDataset(const string &filename, const string &groupname, const string &datasetname, const float *data, const uint2 &dim);
template void cuDFNsys::HDF5API::AddDataset(const string &filename, const string &groupname, const string &datasetname, const size_t *data, const uint2 &dim);
template void cuDFNsys::HDF5API::AddDataset(const string &filename, const string &groupname, const string &datasetname, const uint *data, const uint2 &dim);

// ====================================================
// NAME:        AddDataset
// DESCRIPTION: add a number of datasets with a group
//              name to a h5 file
// AUTHOR:      Tingchang YIN
// DATE:        09/04/2022
// ====================================================
template <class T>
void cuDFNsys::HDF5API::AddDatasetsWithOneGroup(const string &filename,
                                                const string &groupname,
                                                const vector<string> &datasetname,
                                                const vector<T *> data,
                                                const vector<uint2> &dim)
{
    H5File file(filename, H5F_ACC_RDWR);

    Group group;

    H5::Exception::dontPrint();
    try
    {
        //cout << "try to open a group!\n";
        group = file.openGroup(groupname);
        //cout << "opened group!\n";
    }
    catch (...)
    {
        //cout << "no this group! create a new group!\n";
        group = file.createGroup(groupname);
        //cout << "created group!\n";
    };

    for (int i = 0; i < datasetname.size(); ++i)
    {
        hsize_t *dims; // dataset dimensions for each rank
        int rank_ = 1;
        if (dim[i].y > 1)
            rank_ = 2;

        dims = new hsize_t[rank_];
        dims[0] = dim[i].x;
        if (rank_ == 2)
            dims[1] = dim[i].y;
        DataSpace dataspace(rank_, dims);
        delete[] dims;
        dims = NULL;

        DataSet dataset =
            group.createDataSet(datasetname[i], PredType::NATIVE_DOUBLE, dataspace);

        double *buffer = new double[dim[i].x * dim[i].y]();
        if (buffer == NULL)
        {
            string AS = "Alloc error in HDF5API::AddDatasetsWithOneGroup!\n";
            group.close();
            file.close();
            throw ExceptionsPause(AS);
        }

        for (size_t j = 0; j < dim[i].x * dim[i].y; ++j)
            buffer[j] = data[i][j];

        dataset.write(buffer, PredType::NATIVE_DOUBLE);

        delete[] buffer;
        buffer = NULL;
    }

    group.close();
    file.close();
}; // AddDatasetsWithOneGroup
template void cuDFNsys::HDF5API::AddDatasetsWithOneGroup(const string &filename, const string &groupname, const vector<string> &datasetname, const vector<int *> data, const vector<uint2> &dim);
template void cuDFNsys::HDF5API::AddDatasetsWithOneGroup(const string &filename, const string &groupname, const vector<string> &datasetname, const vector<double *> data, const vector<uint2> &dim);
template void cuDFNsys::HDF5API::AddDatasetsWithOneGroup(const string &filename, const string &groupname, const vector<string> &datasetname, const vector<float *> data, const vector<uint2> &dim);
template void cuDFNsys::HDF5API::AddDatasetsWithOneGroup(const string &filename, const string &groupname, const vector<string> &datasetname, const vector<size_t *> data, const vector<uint2> &dim);
template void cuDFNsys::HDF5API::AddDatasetsWithOneGroup(const string &filename, const string &groupname, const vector<string> &datasetname, const vector<uint *> data, const vector<uint2> &dim);

// ====================================================
// NAME:        OverWrite
// DESCRIPTION: OverWrite dataset to a h5
// AUTHOR:      Tingchang YIN
// DATE:        09/04/2022
// ====================================================
template <class T>
void cuDFNsys::HDF5API::OverWrite(const string &filename,
                                  const string &groupname,
                                  const string &datasetname,
                                  const T *data,
                                  const uint2 &dim)
{
    H5File file(filename, H5F_ACC_RDWR); //The hdf5 c++ object.

    string channelName;
    if (groupname != "N")
        channelName = "/" + groupname + "/" + datasetname;
    else
        channelName = "/" + datasetname;

    int result = H5Ldelete(file.getId(), channelName.data(), H5P_DEFAULT);
    result++;
    file.close();

    this->AddDataset(filename, groupname,
                     datasetname, data, dim);
}; // OverWrite
template void cuDFNsys::HDF5API::OverWrite(const string &filename, const string &groupname, const string &datasetname, const int *data, const uint2 &dim);
template void cuDFNsys::HDF5API::OverWrite(const string &filename, const string &groupname, const string &datasetname, const double *data, const uint2 &dim);
template void cuDFNsys::HDF5API::OverWrite(const string &filename, const string &groupname, const string &datasetname, const float *data, const uint2 &dim);
template void cuDFNsys::HDF5API::OverWrite(const string &filename, const string &groupname, const string &datasetname, const size_t *data, const uint2 &dim);
template void cuDFNsys::HDF5API::OverWrite(const string &filename, const string &groupname, const string &datasetname, const uint *data, const uint2 &dim);

// ====================================================
// NAME:        ReadDataset
// DESCRIPTION: Read dataset in a h5
// AUTHOR:      Tingchang YIN
// DATE:        09/04/2022
// ====================================================
vector<double> cuDFNsys::HDF5API::ReadDataset(const string &filename,
                                              const string &groupname,
                                              const string &datasetname)
{
    try
    {
        H5::Exception::dontPrint();
        H5File file1(filename, H5F_ACC_RDONLY);
        file1.close();
    }
    catch (...)
    {
        string AS = "File '" + filename + "' does not exist!\n";
        throw ExceptionsPause(AS);
    };

    H5File file(filename, H5F_ACC_RDONLY);
    DataSet dataset;
    Group group;
    if (groupname != "N")
    {
        Group group = file.openGroup(groupname);
        dataset = group.openDataSet(datasetname);
    }
    else
        dataset = file.openDataSet(datasetname);

    DataSpace filespace = dataset.getSpace();
    int rank = filespace.getSimpleExtentNdims();

    hsize_t dims[rank];

    rank = filespace.getSimpleExtentDims(dims);

    DataSpace myspace(rank, dims);

    int NUM_size = 1;
    for (int i = 0; i < rank; ++i)
        NUM_size *= dims[i];

    double *buffer = new double[NUM_size]();
    if (buffer == NULL)
    {
        string AS = "Alloc error in HDF5API::ReadDataset!\n";
        throw ExceptionsPause(AS);
    }

    dataset.read(buffer, PredType::NATIVE_DOUBLE, myspace, filespace);

    vector<double> AK(buffer, buffer + NUM_size);

    delete[] buffer;
    buffer = NULL;
    if (groupname != "N")
        group.close();
    file.close();

    return AK;
}; // ReadDataset

// ====================================================
// NAME:        IfH5FileExist
// DESCRIPTION: check if a h5 exists?
// AUTHOR:      Tingchang YIN
// DATE:        09/04/2022
// ====================================================
bool cuDFNsys::HDF5API::IfH5FileExist(const string &filename)
{
    try
    {
        H5::Exception::dontPrint();
        H5File file(filename, H5F_ACC_RDONLY);
        file.close();
        return true;
    }
    catch (...)
    {
        return false;
    };
    return false;
}; //IfH5FileExist