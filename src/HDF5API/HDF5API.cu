#include "hip/hip_runtime.h"
/****************************************************************************
* cuDFNsys - simulating flow and transport in 3D fracture networks          *
* Copyright (C) 2022, Tingchang YIN, Sergio GALINDO-TORRES                  *
*                                                                           *
* This program is free software: you can redistribute it and/or modify      *
* it under the terms of the GNU Affero General Public License as            *
* published by the Free Software Foundation, either version 3 of the        *
* License, or (at your option) any later version.                           *
*                                                                           *
* This program is distributed in the hope that it will be useful,           *
* but WITHOUT ANY WARRANTY; without even the implied warranty of            *
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the             *
* GNU Affero General Public License for more details.                       *
*                                                                           *
* You should have received a copy of the GNU Affero General Public License  *
* along with this program.  If not, see <https://www.gnu.org/licenses/>.    *
*****************************************************************************/

#include "HDF5API/HDF5API.cuh"

// ====================================================
// NAME:        NewFile
// DESCRIPTION: create a new h5 file
// AUTHOR:      Tingchang YIN
// DATE:        09/04/2022
// ====================================================
void cuDFNsys::HDF5API::NewFile(const string &name)
{
    try
    {
        H5::Exception::dontPrint();
        H5File file(name, H5F_ACC_TRUNC);
        file.close();
    }
    catch (...)
    {
        cout << "\033[31mA file with same name does exist and is occupied by "
                "anthor program now!\033[0m\n";
        cout << "\033[31mSo, I will delete this file and create a new "
                "one!\033[0m\n";
        string name1 = name;
        std::remove(name1.c_str());
        H5File file(name, H5F_ACC_TRUNC);
        file.close();
    }
}; // NewFile

// ====================================================
// NAME:        AddDataset
// DESCRIPTION: add dataset to a h5 file
// AUTHOR:      Tingchang YIN
// DATE:        09/04/2022
// ====================================================
template <class T>
void cuDFNsys::HDF5API::AddDataset(
    const string &filename, const string &groupname, const string &datasetname,
    const T *data, // the T should be column-major
    const uint2 &dim)
{
    H5File file(filename, H5F_ACC_RDWR);

    hsize_t *dims; // dataset dimensions for each rank
    int rank_ = 1;
    if (dim.y > 1)
        rank_ = 2;

    dims = new hsize_t[2];
    dims[0] = dim.x;
    if (rank_ == 2)
        dims[1] = dim.y;
    //cout << dims[0] << ", " << dims[1] << endl;

    DataSpace dataspace;

    if (rank_ == 1)
        dataspace.setExtentSimple(rank_, &dims[0]);
    else
        dataspace.setExtentSimple(rank_, dims);

    delete[] dims;
    dims = NULL;

    auto datatype = PredType::NATIVE_DOUBLE;

    if (typeid(data[0]) == typeid(double))
        datatype = PredType::NATIVE_DOUBLE;
    else if (typeid(data[0]) == typeid(float))
        datatype = PredType::NATIVE_FLOAT;
    else if (typeid(data[0]) == typeid(size_t))
    {
        string kklo = "In cuDFNsys::HDF5API, writing size_t data might lead to "
                      "problems, please cast size_t array to int/uint array\n";
        cout << kklo;
        throw ExceptionsPause(kklo);
    }
    else if (typeid(data[0]) == typeid(int))
        datatype = PredType::NATIVE_INT;
    else if (typeid(data[0]) == typeid(uint))
        datatype = PredType::NATIVE_UINT;
    else
        throw ExceptionsPause("Undefined datatype in HDF5API::AddDataset\n");

    if (groupname != "N")
    {
        Group group;

        H5::Exception::dontPrint();
        try
        {
            //cout << "try to open a group!\n";
            group = file.openGroup(groupname);
            //cout << "opened group!\n";
        }
        catch (...)
        {
            //cout << "no this group! create a new group!\n";
            group = file.createGroup(groupname);
            //cout << "created group!\n";
        }

        DataSet dataset = group.createDataSet(datasetname, datatype, dataspace);

        dataset.write(data, datatype);

        group.close();
    }
    else
    {
        DataSet dataset = file.createDataSet(datasetname, datatype, dataspace);

        dataset.write(data, datatype);
    }

    file.close();
}; // AddDataset
template void cuDFNsys::HDF5API::AddDataset(const string &filename,
                                            const string &groupname,
                                            const string &datasetname,
                                            const int *data, const uint2 &dim);
template void cuDFNsys::HDF5API::AddDataset(const string &filename,
                                            const string &groupname,
                                            const string &datasetname,
                                            const double *data,
                                            const uint2 &dim);
template void cuDFNsys::HDF5API::AddDataset(const string &filename,
                                            const string &groupname,
                                            const string &datasetname,
                                            const float *data,
                                            const uint2 &dim);
template void cuDFNsys::HDF5API::AddDataset(const string &filename,
                                            const string &groupname,
                                            const string &datasetname,
                                            const size_t *data,
                                            const uint2 &dim);
template void cuDFNsys::HDF5API::AddDataset(const string &filename,
                                            const string &groupname,
                                            const string &datasetname,
                                            const uint *data, const uint2 &dim);

// ====================================================
// NAME:        AddDataset
// DESCRIPTION: add a number of datasets with a group
//              name to a h5 file
// AUTHOR:      Tingchang YIN
// DATE:        09/04/2022
// ====================================================
template <class T>
void cuDFNsys::HDF5API::AddDatasetsWithOneGroup(
    const string &filename, const string &groupname,
    const vector<string> &datasetname, const vector<T *> data,
    const vector<uint2> &dim)
{
    // if (groupname == "N")
    //     throw ExceptionsPause("You should define group name when you are using "
    //                           "HDF5API::AddDatasetsWithOneGroup!\n");

    if (groupname != "N")
    {
        H5File file(filename, H5F_ACC_RDWR);

        Group group;

        H5::Exception::dontPrint();
        try
        {

            //cout << "try to open a group!\n";
            group = file.openGroup(groupname);
            //cout << "opened group!\n";
        }
        catch (...)
        {
            //cout << "no this group! create a new group!\n";
            group = file.createGroup(groupname);
            //cout << "created group!\n";
        };

        auto datatype = PredType::NATIVE_DOUBLE;

        if (typeid(data[0][0]) == typeid(double))
            datatype = PredType::NATIVE_DOUBLE;
        else if (typeid(data[0][0]) == typeid(float))
            datatype = PredType::NATIVE_FLOAT;
        else if (typeid(data[0][0]) == typeid(size_t))
        {
            string kklo =
                "In cuDFNsys::HDF5API, writing size_t data might lead to "
                "problems, please cast size_t array to int/uint array\n";
            cout << kklo;
            throw ExceptionsPause(kklo);
        }
        else if (typeid(data[0][0]) == typeid(int))
            datatype = PredType::NATIVE_INT;
        else if (typeid(data[0][0]) == typeid(uint))
            datatype = PredType::NATIVE_UINT;
        else
            throw ExceptionsPause(
                "Undefined datatype in HDF5API::AddDataset\n");

        for (int i = 0; i < datasetname.size(); ++i)
        {
            hsize_t *dims; // dataset dimensions for each rank
            int rank_ = 1;
            if (dim[i].y > 1)
                rank_ = 2;

            dims = new hsize_t[rank_];
            dims[0] = dim[i].x;
            if (rank_ == 2)
                dims[1] = dim[i].y;
            DataSpace dataspace(rank_, dims);
            delete[] dims;
            dims = NULL;

            DataSet dataset;

            dataset = group.createDataSet(datasetname[i], datatype, dataspace);

            dataset.write(data[i], datatype);
        }

        group.close();
        file.close();
    }
    else
    {
        H5File file(filename, H5F_ACC_RDWR);

        auto datatype = PredType::NATIVE_DOUBLE;

        if (typeid(data[0][0]) == typeid(double))
            datatype = PredType::NATIVE_DOUBLE;
        else if (typeid(data[0][0]) == typeid(float))
            datatype = PredType::NATIVE_FLOAT;
        else if (typeid(data[0][0]) == typeid(size_t))
        {
            string kklo =
                "In cuDFNsys::HDF5API, writing size_t data might lead to "
                "problems, please cast size_t array to int/uint array\n";
            cout << kklo;
            throw ExceptionsPause(kklo);
        }
        else if (typeid(data[0][0]) == typeid(int))
            datatype = PredType::NATIVE_INT;
        else if (typeid(data[0][0]) == typeid(uint))
            datatype = PredType::NATIVE_UINT;
        else
            throw ExceptionsPause(
                "Undefined datatype in HDF5API::AddDataset\n");

        for (int i = 0; i < datasetname.size(); ++i)
        {
            /// cout << "i: " << i << ", " << data[i][0] << ", " << data[i][1]
            ///      << ", " << data[i][2] << endl;
            hsize_t *dims; // dataset dimensions for each rank
            int rank_ = 1;
            if (dim[i].y > 1)
                rank_ = 2;

            dims = new hsize_t[rank_];
            dims[0] = dim[i].x;
            if (rank_ == 2)
                dims[1] = dim[i].y;
            DataSpace dataspace(rank_, dims);

            // for (int i = 0; i < rank_; ++i)
            //     cout << "data size: " << dims[i] << endl;

            delete[] dims;
            dims = NULL;

            //cout << "dataseru\n";
            DataSet dataset =
                file.createDataSet(datasetname[i], datatype, dataspace);

            dataset.write(data[i], datatype);

            //cout << "written\n";
        }
        file.close();
    }
}; // AddDatasetsWithOneGroup
template void cuDFNsys::HDF5API::AddDatasetsWithOneGroup(
    const string &filename, const string &groupname,
    const vector<string> &datasetname, const vector<int *> data,
    const vector<uint2> &dim);
template void cuDFNsys::HDF5API::AddDatasetsWithOneGroup(
    const string &filename, const string &groupname,
    const vector<string> &datasetname, const vector<double *> data,
    const vector<uint2> &dim);
template void cuDFNsys::HDF5API::AddDatasetsWithOneGroup(
    const string &filename, const string &groupname,
    const vector<string> &datasetname, const vector<float *> data,
    const vector<uint2> &dim);
template void cuDFNsys::HDF5API::AddDatasetsWithOneGroup(
    const string &filename, const string &groupname,
    const vector<string> &datasetname, const vector<size_t *> data,
    const vector<uint2> &dim);
template void cuDFNsys::HDF5API::AddDatasetsWithOneGroup(
    const string &filename, const string &groupname,
    const vector<string> &datasetname, const vector<uint *> data,
    const vector<uint2> &dim);

// ====================================================
// NAME:        OverWrite
// DESCRIPTION: OverWrite dataset to a h5
// AUTHOR:      Tingchang YIN
// DATE:        09/04/2022
// ====================================================
template <class T>
void cuDFNsys::HDF5API::OverWrite(const string &filename,
                                  const string &groupname,
                                  const string &datasetname, const T *data,
                                  const uint2 &dim)
{
    H5File file(filename, H5F_ACC_RDWR); //The hdf5 c++ object.

    string channelName;
    if (groupname != "N")
        channelName = "/" + groupname + "/" + datasetname;
    else
        channelName = "/" + datasetname;

    int result = H5Ldelete(file.getId(), channelName.data(), H5P_DEFAULT);
    result++;
    file.close();

    this->AddDataset(filename, groupname, datasetname, data, dim);
}; // OverWrite
template void cuDFNsys::HDF5API::OverWrite(const string &filename,
                                           const string &groupname,
                                           const string &datasetname,
                                           const int *data, const uint2 &dim);
template void cuDFNsys::HDF5API::OverWrite(const string &filename,
                                           const string &groupname,
                                           const string &datasetname,
                                           const double *data,
                                           const uint2 &dim);
template void cuDFNsys::HDF5API::OverWrite(const string &filename,
                                           const string &groupname,
                                           const string &datasetname,
                                           const float *data, const uint2 &dim);
template void cuDFNsys::HDF5API::OverWrite(const string &filename,
                                           const string &groupname,
                                           const string &datasetname,
                                           const size_t *data,
                                           const uint2 &dim);
template void cuDFNsys::HDF5API::OverWrite(const string &filename,
                                           const string &groupname,
                                           const string &datasetname,
                                           const uint *data, const uint2 &dim);

// ====================================================
// NAME:        ReadDataset
// DESCRIPTION: Read dataset in a h5
// AUTHOR:      Tingchang YIN
// DATE:        09/04/2022
// ====================================================
template <class T>
vector<T> cuDFNsys::HDF5API::ReadDataset(const string &filename,
                                         const string &groupname,
                                         const string &datasetname)
{
    try
    {
        H5::Exception::dontPrint();
        H5File file1(filename, H5F_ACC_RDONLY);
        file1.close();
    }
    catch (...)
    {
        string AS = "File '" + filename + "' does not exist!\n";
        throw ExceptionsPause(AS);
    };

    H5File file(filename, H5F_ACC_RDONLY);
    DataSet dataset;
    Group group;
    if (groupname != "N")
    {
        Group group = file.openGroup(groupname);
        dataset = group.openDataSet(datasetname);
    }
    else
        dataset = file.openDataSet(datasetname);

    DataSpace filespace = dataset.getSpace();
    int rank = filespace.getSimpleExtentNdims();

    DataType dt_org = dataset.getDataType();

    hsize_t dims[rank];

    rank = filespace.getSimpleExtentDims(dims);

    DataSpace myspace(rank, dims);

    int NUM_size = 1;
    for (int i = 0; i < rank; ++i)
        NUM_size *= dims[i];

    T *buffer = new T[NUM_size];

    if (typeid(buffer[0]) == typeid(double))
        dataset.read(buffer, PredType::NATIVE_DOUBLE, myspace, filespace);
    else if (typeid(buffer[0]) == typeid(float))
        dataset.read(buffer, PredType::NATIVE_FLOAT, myspace, filespace);
    else if (typeid(buffer[0]) == typeid(int))
        dataset.read(buffer, PredType::NATIVE_INT, myspace, filespace);
    else if (typeid(buffer[0]) == typeid(uint))
        dataset.read(buffer, PredType::NATIVE_UINT, myspace, filespace);
    else
        throw ExceptionsPause("Undefined datatype in HDF5API::ReadDataset\n");

    vector<T> AK(buffer, buffer + NUM_size);

    //cout << AK[0] << endl;

    delete[] buffer;
    buffer = NULL;

    if (groupname != "N")
        group.close();
    file.close();

    return AK;
}; // ReadDataset
template vector<double> cuDFNsys::HDF5API::ReadDataset<double>(
    const string &filename, const string &groupname, const string &datasetname);
template vector<float> cuDFNsys::HDF5API::ReadDataset<float>(
    const string &filename, const string &groupname, const string &datasetname);
template vector<uint> cuDFNsys::HDF5API::ReadDataset<uint>(
    const string &filename, const string &groupname, const string &datasetname);
template vector<int> cuDFNsys::HDF5API::ReadDataset<int>(
    const string &filename, const string &groupname, const string &datasetname);

// ====================================================
// NAME:        IfH5FileExist
// DESCRIPTION: check if a h5 exists?
// AUTHOR:      Tingchang YIN
// DATE:        09/04/2022
// ====================================================
bool cuDFNsys::HDF5API::IfH5FileExist(const string &filename)
{
    try
    {
        H5::Exception::dontPrint();
        H5File file(filename, H5F_ACC_RDONLY);
        file.close();
        return true;
    }
    catch (...)
    {
        return false;
    };
    return false;
}; //IfH5FileExist

// ====================================================
// NAME:        AddDatasetString
// DESCRIPTION: add string to a file
// AUTHOR:      Tingchang YIN
// DATE:        21/08/2022
// ====================================================
void cuDFNsys::HDF5API::AddDatasetString(
    const string &filename, const string &groupname, const string &datasetname,
    const string &sdata) // the T should be column-major

{
    H5File file(filename, H5F_ACC_RDWR);

    H5::StrType datatype(H5::PredType::C_S1, sdata.length() + 1);

    if (groupname != "N")
    {
        Group group;

        H5::Exception::dontPrint();
        try
        {
            //cout << "try to open a group!\n";
            group = file.openGroup(groupname);
            //cout << "opened group!\n";
        }
        catch (...)
        {
            //cout << "no this group! create a new group!\n";
            group = file.createGroup(groupname);
            //cout << "created group!\n";
        }

        DataSet dataset = group.createDataSet(datasetname, datatype,
                                              H5::DataSpace(H5S_SCALAR));

        //char *buffer = new double[dim.x * dim.y]();
        dataset.write(sdata.data(), datatype);

        //delete[] buffer;
        //buffer = NULL;

        group.close();
    }
    else
    {

        DataSet dataset = file.createDataSet(datasetname, datatype,
                                             H5::DataSpace(H5S_SCALAR));

        //const char *buffer = sdata.data();

        dataset.write(sdata.data(), datatype);
    }

    file.close();
}; // AddDatasetString

// ====================================================
// NAME:        ReadDatasetString
// DESCRIPTION: read string in a file
// AUTHOR:      Tingchang YIN
// DATE:        22/08/2022
// ====================================================
string cuDFNsys::HDF5API::ReadDatasetString(const string &filename,
                                            const string &groupname,
                                            const string &datasetname)
{
    try
    {
        H5::Exception::dontPrint();
        H5File file1(filename, H5F_ACC_RDONLY);
        file1.close();
    }
    catch (...)
    {
        string AS = "File '" + filename + "' does not exist!\n";
        throw ExceptionsPause(AS);
    };

    H5File file(filename, H5F_ACC_RDONLY);
    DataSet dataset;
    Group group;
    if (groupname != "N")
    {
        Group group = file.openGroup(groupname);
        dataset = group.openDataSet(datasetname);
    }
    else
        dataset = file.openDataSet(datasetname);

    DataSpace filespace = dataset.getSpace();
    H5::StrType datatype = dataset.getStrType();

    std::string data;

    dataset.read(data, datatype, filespace);

    if (groupname != "N")
        group.close();
    file.close();

    return data;
}; // ReadDatasetString
