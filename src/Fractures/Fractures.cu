#include "hip/hip_runtime.h"
#include "Fractures/Fractures.cuh"

// ====================================================
// NAME:        Fractures
// DESCRIPTION: Fractures in a DFN
// AUTHOR:      Tingchang YIN
// DATE:        04/04/2022
// ====================================================
__global__ void cuDFNsys::Fractures(cuDFNsys::Fracture *verts,
                                    unsigned long seed,
                                    int count,
                                    float model_L,
                                    uint ModeSizeDistri,
                                    float4 ParaSizeDistri,
                                    float kappa,
                                    float conductivity_powerlaw_exponent)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i > count - 1)
        return;

    hiprandState state;

    hiprand_init(seed, i, 0, &state);

    float R_ = 0;

    // if (alpha == 0 && abs(minR - maxR) < 1e-7)
    //     R_ = minR;
    // else if (alpha == 0 && abs(minR - maxR) > 1e-7)
    //     R_ = cuDFNsys::RandomUniform(minR, maxR, hiprand_uniform(&state));
    // else
    //     R_ = cuDFNsys::RandomPowerlaw(minR, maxR, alpha, hiprand_uniform(&state));

    if (ModeSizeDistri == 0)
        R_ = cuDFNsys::RandomPowerlaw(ParaSizeDistri.y, ParaSizeDistri.z,
                                      ParaSizeDistri.x, hiprand_uniform(&state));
    else if (ModeSizeDistri == 1)
        R_ = cuDFNsys::RandomLognormal(ParaSizeDistri.x,
                                       ParaSizeDistri.y,
                                       ParaSizeDistri.z,
                                       ParaSizeDistri.w, hiprand_uniform(&state));
    else if (ModeSizeDistri == 2)
        R_ = cuDFNsys::RandomUniform(ParaSizeDistri.x,
                                     ParaSizeDistri.y, hiprand_uniform(&state));
    else if (ModeSizeDistri == 3)
        R_ = ParaSizeDistri.x;

    verts[i].Radius = R_;
    //printf("%f\n", verts[i].Radius);

    verts[i].NumVertsTruncated = 4;
    //printf("conductivity_powerlaw_exponent: %f, kappa: %f\n", conductivity_powerlaw_exponent, kappa);

    if (conductivity_powerlaw_exponent == 0)
        verts[i].Conductivity = 1.0;
    else
        verts[i].Conductivity = (1.0e-11) * pow(R_, 3.0 * conductivity_powerlaw_exponent);

    verts[i].Center.x = cuDFNsys::RandomUniform(-model_L * 0.5, model_L * 0.5,
                                                hiprand_uniform(&state));
    verts[i].Center.y = cuDFNsys::RandomUniform(-model_L * 0.5, model_L * 0.5,
                                                hiprand_uniform(&state));
    verts[i].Center.z = cuDFNsys::RandomUniform(-model_L * 0.5, model_L * 0.5,
                                                hiprand_uniform(&state));

    verts[i].NormalVec = make_float3(cuDFNsys::RandomUniform(-1.0, 1.0, hiprand_uniform(&state)),
                                     cuDFNsys::RandomUniform(-1.0, 1.0, hiprand_uniform(&state)),
                                     0);
    float R_xy = sqrt(verts[i].NormalVec.x * verts[i].NormalVec.x +
                      verts[i].NormalVec.y * verts[i].NormalVec.y);
    verts[i].NormalVec.z = R_xy / tan(cuDFNsys::RandomFisher(hiprand_uniform(&state), kappa));
    float norm_f = sqrt(verts[i].NormalVec.x * verts[i].NormalVec.x +
                        verts[i].NormalVec.y * verts[i].NormalVec.y +
                        verts[i].NormalVec.z * verts[i].NormalVec.z);
    verts[i].NormalVec.x /= norm_f;
    verts[i].NormalVec.y /= norm_f;
    verts[i].NormalVec.z /= norm_f;

    float *normal_fff = &verts[i].NormalVec.x;
    float *verts_3D_ptr = &verts[i].Verts3D[0].x;
    for (int j = 0; j < 3; ++j)
    {
        if (abs(normal_fff[j]) > 1e-3)
        {
            verts_3D_ptr[(j + 1) % 3] = cuDFNsys::RandomUniform(-1.0, 1.0, hiprand_uniform(&state));
            verts_3D_ptr[(j + 2) % 3] = cuDFNsys::RandomUniform(-1.0, 1.0, hiprand_uniform(&state));
            verts_3D_ptr[j] = -1.0 * (verts_3D_ptr[(j + 1) % 3] * normal_fff[(j + 1) % 3] + verts_3D_ptr[(j + 2) % 3] * normal_fff[(j + 2) % 3]) / normal_fff[j];
            break;
        }
    }

    float norm_vert1 = sqrt(verts[i].Verts3D[0].x * verts[i].Verts3D[0].x +
                            verts[i].Verts3D[0].y * verts[i].Verts3D[0].y +
                            verts[i].Verts3D[0].z * verts[i].Verts3D[0].z);
    norm_vert1 = R_ / norm_vert1;
    verts[i].Verts3D[0].x *= norm_vert1;
    verts[i].Verts3D[0].y *= norm_vert1;
    verts[i].Verts3D[0].z *= norm_vert1;
    verts[i].Verts3D[2].x = -1.0 * verts[i].Verts3D[0].x;
    verts[i].Verts3D[2].y = -1.0 * verts[i].Verts3D[0].y;
    verts[i].Verts3D[2].z = -1.0 * verts[i].Verts3D[0].z;

    verts[i].Verts3D[1] = cuDFNsys::CrossProductFloat3(verts[i].NormalVec,
                                                       verts[i].Verts3D[0]);
    norm_vert1 = sqrt(verts[i].Verts3D[1].x * verts[i].Verts3D[1].x +
                      verts[i].Verts3D[1].y * verts[i].Verts3D[1].y +
                      verts[i].Verts3D[1].z * verts[i].Verts3D[1].z);
    norm_vert1 = R_ / norm_vert1;
    verts[i].Verts3D[1].x *= norm_vert1;
    verts[i].Verts3D[1].y *= norm_vert1;
    verts[i].Verts3D[1].z *= norm_vert1;
    verts[i].Verts3D[3].x = -1.0 * verts[i].Verts3D[1].x;
    verts[i].Verts3D[3].y = -1.0 * verts[i].Verts3D[1].y;
    verts[i].Verts3D[3].z = -1.0 * verts[i].Verts3D[1].z;
    //-----------------------------------------
    for (int j = 0; j < 4; ++j)
    {
        verts[i].Verts3D[j].x += verts[i].Center.x;
        verts[i].Verts3D[j].y += verts[i].Center.y;
        verts[i].Verts3D[j].z += verts[i].Center.z;

        verts[i].Verts3DTruncated[j].x = verts[i].Verts3D[j].x;
        verts[i].Verts3DTruncated[j].y = verts[i].Verts3D[j].y;
        verts[i].Verts3DTruncated[j].z = verts[i].Verts3D[j].z;
    };

    bool gh = cuDFNsys::TruncateFracture(&verts[i], model_L, 0, 1);
    verts[i].ConnectModelSurf[0] = gh;

    gh = cuDFNsys::TruncateFracture(&verts[i], model_L, 0, -1);
    verts[i].ConnectModelSurf[1] = gh;

    gh = cuDFNsys::TruncateFracture(&verts[i], model_L, 1, 1);
    verts[i].ConnectModelSurf[2] = gh;

    gh = cuDFNsys::TruncateFracture(&verts[i], model_L, 1, -1);
    verts[i].ConnectModelSurf[3] = gh;

    gh = cuDFNsys::TruncateFracture(&verts[i], model_L, 2, 1);
    verts[i].ConnectModelSurf[4] = gh;

    gh = cuDFNsys::TruncateFracture(&verts[i], model_L, 2, -1);
    verts[i].ConnectModelSurf[5] = gh;
}; // Fractures

// ====================================================
// NAME:        FracturesCrossedVertical
// DESCRIPTION: two crossed vertical fractures
// AUTHOR:      Tingchang YIN
// DATE:        20/04/2022
// ====================================================
__global__ void cuDFNsys::FracturesCrossedVertical(cuDFNsys::Fracture *verts,
                                                   unsigned long seed,
                                                   int count,
                                                   float model_L)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i >= 2)
        return;

    hiprandState state;

    hiprand_init(seed, i, 0, &state);

    float R_ = sqrt(model_L * model_L / 2.0f);

    verts[i].Radius = R_;
    //printf("%f\n", verts[i].Radius);

    verts[i].NumVertsTruncated = 4;
    //printf("conductivity_powerlaw_exponent: %f, kappa: %f\n", conductivity_powerlaw_exponent, kappa);

    verts[i].Conductivity = 1;

    verts[i].Center.x = 0;
    verts[i].Center.y = 0;
    verts[i].Center.z = 0;

    if (i == 0)
        verts[i].NormalVec = make_float3(0, 1.0f, 0);
    else
        verts[i].NormalVec = make_float3(1.0f, 0, 0);

    if (i == 0)
        verts[i].Verts3D[0] = make_float3(-0.5 * model_L, 0, 0.5 * model_L);
    else
        verts[i].Verts3D[0] = make_float3(0, -0.5 * model_L, 0.5 * model_L);

    float norm_vert1 = sqrt(verts[i].Verts3D[0].x * verts[i].Verts3D[0].x +
                            verts[i].Verts3D[0].y * verts[i].Verts3D[0].y +
                            verts[i].Verts3D[0].z * verts[i].Verts3D[0].z);
    norm_vert1 = R_ / norm_vert1;
    verts[i].Verts3D[0].x *= norm_vert1;
    verts[i].Verts3D[0].y *= norm_vert1;
    verts[i].Verts3D[0].z *= norm_vert1;
    verts[i].Verts3D[2].x = -1.0 * verts[i].Verts3D[0].x;
    verts[i].Verts3D[2].y = -1.0 * verts[i].Verts3D[0].y;
    verts[i].Verts3D[2].z = -1.0 * verts[i].Verts3D[0].z;

    verts[i].Verts3D[1] = cuDFNsys::CrossProductFloat3(verts[i].NormalVec,
                                                       verts[i].Verts3D[0]);
    norm_vert1 = sqrt(verts[i].Verts3D[1].x * verts[i].Verts3D[1].x +
                      verts[i].Verts3D[1].y * verts[i].Verts3D[1].y +
                      verts[i].Verts3D[1].z * verts[i].Verts3D[1].z);
    norm_vert1 = R_ / norm_vert1;
    verts[i].Verts3D[1].x *= norm_vert1;
    verts[i].Verts3D[1].y *= norm_vert1;
    verts[i].Verts3D[1].z *= norm_vert1;
    verts[i].Verts3D[3].x = -1.0 * verts[i].Verts3D[1].x;
    verts[i].Verts3D[3].y = -1.0 * verts[i].Verts3D[1].y;
    verts[i].Verts3D[3].z = -1.0 * verts[i].Verts3D[1].z;
    //-----------------------------------------
    for (int j = 0; j < 4; ++j)
    {
        verts[i].Verts3D[j].x += verts[i].Center.x;
        verts[i].Verts3D[j].y += verts[i].Center.y;
        verts[i].Verts3D[j].z += verts[i].Center.z;

        verts[i].Verts3DTruncated[j].x = verts[i].Verts3D[j].x;
        verts[i].Verts3DTruncated[j].y = verts[i].Verts3D[j].y;
        verts[i].Verts3DTruncated[j].z = verts[i].Verts3D[j].z;
    };

    bool gh = cuDFNsys::TruncateFracture(&verts[i], model_L, 0, 1);
    verts[i].ConnectModelSurf[0] = gh;

    gh = cuDFNsys::TruncateFracture(&verts[i], model_L, 0, -1);
    verts[i].ConnectModelSurf[1] = gh;

    gh = cuDFNsys::TruncateFracture(&verts[i], model_L, 1, 1);
    verts[i].ConnectModelSurf[2] = gh;

    gh = cuDFNsys::TruncateFracture(&verts[i], model_L, 1, -1);
    verts[i].ConnectModelSurf[3] = gh;

    gh = cuDFNsys::TruncateFracture(&verts[i], model_L, 2, 1);
    verts[i].ConnectModelSurf[4] = gh;

    gh = cuDFNsys::TruncateFracture(&verts[i], model_L, 2, -1);
    verts[i].ConnectModelSurf[5] = gh;
}; // FracturesCrossedVertical

// ====================================================
// NAME:        FracturesBeta50Beta60
// DESCRIPTION: two inclined fractures, with two beta values
// AUTHOR:      Tingchang YIN
// DATE:        20/04/2022
// ====================================================
__global__ void cuDFNsys::FracturesBeta50Beta60(cuDFNsys::Fracture *verts,
                                                unsigned long seed,
                                                int count,
                                                float model_L)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i >= 2)
        return;

    hiprandState state;

    hiprand_init(seed, i, 0, &state);

    float R_ = model_L * 3.0f;

    verts[i].Radius = R_;
    //printf("%f\n", verts[i].Radius);

    verts[i].NumVertsTruncated = 4;
    //printf("conductivity_powerlaw_exponent: %f, kappa: %f\n", conductivity_powerlaw_exponent, kappa);

    verts[i].Conductivity = 1.0;

    if (i == 0)
        verts[i].NormalVec = make_float3(1, 0, 0);
    else if (i == 1)
        verts[i].NormalVec = make_float3(-1, 0, 0);

    float R_xy = sqrt(verts[i].NormalVec.x * verts[i].NormalVec.x +
                      verts[i].NormalVec.y * verts[i].NormalVec.y);
    if (i == 0)
        verts[i].NormalVec.z = R_xy / tan(50.0 / 180.0 * M_PI);
    else if (i == 1)
        verts[i].NormalVec.z = R_xy / tan(1.0 / 3.0 * M_PI);
    float norm_f = sqrt(verts[i].NormalVec.x * verts[i].NormalVec.x +
                        verts[i].NormalVec.y * verts[i].NormalVec.y +
                        verts[i].NormalVec.z * verts[i].NormalVec.z);
    verts[i].NormalVec.x /= norm_f;
    verts[i].NormalVec.y /= norm_f;
    verts[i].NormalVec.z /= norm_f;

    verts[i].Center = make_float3(0, 0, 0);

    //----------
    float *normal_fff = &verts[i].NormalVec.x;
    float *verts_3D_ptr = &verts[i].Verts3D[0].x;
    for (int j = 0; j < 3; ++j)
    {
        if (abs(normal_fff[j]) > 1e-3)
        {
            verts_3D_ptr[(j + 1) % 3] = cuDFNsys::RandomUniform(-1.0, 1.0, hiprand_uniform(&state));
            verts_3D_ptr[(j + 2) % 3] = cuDFNsys::RandomUniform(-1.0, 1.0, hiprand_uniform(&state));
            verts_3D_ptr[j] = -1.0 * (verts_3D_ptr[(j + 1) % 3] * normal_fff[(j + 1) % 3] + verts_3D_ptr[(j + 2) % 3] * normal_fff[(j + 2) % 3]) / normal_fff[j];
            break;
        }
    }

    float norm_vert1 = sqrt(verts[i].Verts3D[0].x * verts[i].Verts3D[0].x +
                            verts[i].Verts3D[0].y * verts[i].Verts3D[0].y +
                            verts[i].Verts3D[0].z * verts[i].Verts3D[0].z);
    norm_vert1 = R_ / norm_vert1;
    verts[i].Verts3D[0].x *= norm_vert1;
    verts[i].Verts3D[0].y *= norm_vert1;
    verts[i].Verts3D[0].z *= norm_vert1;
    verts[i].Verts3D[2].x = -1.0 * verts[i].Verts3D[0].x;
    verts[i].Verts3D[2].y = -1.0 * verts[i].Verts3D[0].y;
    verts[i].Verts3D[2].z = -1.0 * verts[i].Verts3D[0].z;

    verts[i].Verts3D[1] = cuDFNsys::CrossProductFloat3(verts[i].NormalVec,
                                                       verts[i].Verts3D[0]);
    norm_vert1 = sqrt(verts[i].Verts3D[1].x * verts[i].Verts3D[1].x +
                      verts[i].Verts3D[1].y * verts[i].Verts3D[1].y +
                      verts[i].Verts3D[1].z * verts[i].Verts3D[1].z);
    norm_vert1 = R_ / norm_vert1;
    verts[i].Verts3D[1].x *= norm_vert1;
    verts[i].Verts3D[1].y *= norm_vert1;
    verts[i].Verts3D[1].z *= norm_vert1;
    verts[i].Verts3D[3].x = -1.0 * verts[i].Verts3D[1].x;
    verts[i].Verts3D[3].y = -1.0 * verts[i].Verts3D[1].y;
    verts[i].Verts3D[3].z = -1.0 * verts[i].Verts3D[1].z;
    //-----------------------------------------
    for (int j = 0; j < 4; ++j)
    {
        verts[i].Verts3D[j].x += verts[i].Center.x;
        verts[i].Verts3D[j].y += verts[i].Center.y;
        verts[i].Verts3D[j].z += verts[i].Center.z;

        verts[i].Verts3DTruncated[j].x = verts[i].Verts3D[j].x;
        verts[i].Verts3DTruncated[j].y = verts[i].Verts3D[j].y;
        verts[i].Verts3DTruncated[j].z = verts[i].Verts3D[j].z;
    };

    bool gh = cuDFNsys::TruncateFracture(&verts[i], model_L, 0, 1);
    verts[i].ConnectModelSurf[0] = gh;

    gh = cuDFNsys::TruncateFracture(&verts[i], model_L, 0, -1);
    verts[i].ConnectModelSurf[1] = gh;

    gh = cuDFNsys::TruncateFracture(&verts[i], model_L, 1, 1);
    verts[i].ConnectModelSurf[2] = gh;

    gh = cuDFNsys::TruncateFracture(&verts[i], model_L, 1, -1);
    verts[i].ConnectModelSurf[3] = gh;

    gh = cuDFNsys::TruncateFracture(&verts[i], model_L, 2, 1);
    verts[i].ConnectModelSurf[4] = gh;

    gh = cuDFNsys::TruncateFracture(&verts[i], model_L, 2, -1);
    verts[i].ConnectModelSurf[5] = gh;
}; // FracturesBeta50Beta60

// ====================================================
// NAME:        FracturesIncomplete
// DESCRIPTION: two incomplet fractures
// AUTHOR:      Tingchang YIN
// DATE:        20/04/2022
// ====================================================
__global__ void cuDFNsys::FracturesIncomplete(cuDFNsys::Fracture *verts,
                                              unsigned long seed,
                                              int count,
                                              float model_L)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i >= 2)
        return;

    hiprandState state;

    hiprand_init(seed, i, 0, &state);

    float R_ = (i == 0 ? 0.55 : 0.60) * model_L;
    ;

    verts[i].Radius = R_;
    //printf("%f\n", verts[i].Radius);

    verts[i].NumVertsTruncated = 4;
    //printf("conductivity_powerlaw_exponent: %f, kappa: %f\n", conductivity_powerlaw_exponent, kappa);

    verts[i].Conductivity = 1.0;

    if (i == 0)
        verts[i].NormalVec = make_float3(1, 0, 0);
    else if (i == 1)
        verts[i].NormalVec = make_float3(0, 1, 0);

    verts[i].Center = make_float3(0, 0, 0);

    //----------
    if (i == 0)
    {
        verts[i].Verts3D[0].y = 0.5 * model_L;
        verts[i].Verts3D[0].x = 0;
        verts[i].Verts3D[0].z = 0;
    }
    else if (i == 1)
    {
        verts[i].Verts3D[0].x = 0.5 * model_L;
        verts[i].Verts3D[0].y = 0;
        verts[i].Verts3D[0].z = 0;
    }

    float norm_vert1 = sqrt(verts[i].Verts3D[0].x * verts[i].Verts3D[0].x +
                            verts[i].Verts3D[0].y * verts[i].Verts3D[0].y +
                            verts[i].Verts3D[0].z * verts[i].Verts3D[0].z);
    norm_vert1 = R_ / norm_vert1;
    verts[i].Verts3D[0].x *= norm_vert1;
    verts[i].Verts3D[0].y *= norm_vert1;
    verts[i].Verts3D[0].z *= norm_vert1;
    verts[i].Verts3D[2].x = -1.0 * verts[i].Verts3D[0].x;
    verts[i].Verts3D[2].y = -1.0 * verts[i].Verts3D[0].y;
    verts[i].Verts3D[2].z = -1.0 * verts[i].Verts3D[0].z;

    verts[i].Verts3D[1] = cuDFNsys::CrossProductFloat3(verts[i].NormalVec,
                                                       verts[i].Verts3D[0]);
    norm_vert1 = sqrt(verts[i].Verts3D[1].x * verts[i].Verts3D[1].x +
                      verts[i].Verts3D[1].y * verts[i].Verts3D[1].y +
                      verts[i].Verts3D[1].z * verts[i].Verts3D[1].z);
    norm_vert1 = R_ / norm_vert1;
    verts[i].Verts3D[1].x *= norm_vert1;
    verts[i].Verts3D[1].y *= norm_vert1;
    verts[i].Verts3D[1].z *= norm_vert1;
    verts[i].Verts3D[3].x = -1.0 * verts[i].Verts3D[1].x;
    verts[i].Verts3D[3].y = -1.0 * verts[i].Verts3D[1].y;
    verts[i].Verts3D[3].z = -1.0 * verts[i].Verts3D[1].z;
    //-----------------------------------------
    for (int j = 0; j < 4; ++j)
    {
        verts[i].Verts3D[j].x += verts[i].Center.x;
        verts[i].Verts3D[j].y += verts[i].Center.y;
        verts[i].Verts3D[j].z += verts[i].Center.z;

        verts[i].Verts3DTruncated[j].x = verts[i].Verts3D[j].x;
        verts[i].Verts3DTruncated[j].y = verts[i].Verts3D[j].y;
        verts[i].Verts3DTruncated[j].z = verts[i].Verts3D[j].z;
    };

    bool gh = cuDFNsys::TruncateFracture(&verts[i], model_L, 0, 1);
    verts[i].ConnectModelSurf[0] = gh;

    gh = cuDFNsys::TruncateFracture(&verts[i], model_L, 0, -1);
    verts[i].ConnectModelSurf[1] = gh;

    gh = cuDFNsys::TruncateFracture(&verts[i], model_L, 1, 1);
    verts[i].ConnectModelSurf[2] = gh;

    gh = cuDFNsys::TruncateFracture(&verts[i], model_L, 1, -1);
    verts[i].ConnectModelSurf[3] = gh;

    gh = cuDFNsys::TruncateFracture(&verts[i], model_L, 2, 1);
    verts[i].ConnectModelSurf[4] = gh;

    gh = cuDFNsys::TruncateFracture(&verts[i], model_L, 2, -1);
    verts[i].ConnectModelSurf[5] = gh;
}; // FracturesIncomplete

// ====================================================
// NAME:        Fractures2DLike
// DESCRIPTION: 2D-like fractures
// AUTHOR:      Tingchang YIN
// DATE:        20/04/2022
// ====================================================
__global__ void cuDFNsys::Fractures2DLike(cuDFNsys::Fracture *verts,
                                          unsigned long seed,
                                          int count,
                                          float model_L,
                                          float alpha,
                                          float minR,
                                          float maxR)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i > count - 1)
        return;

    hiprandState state;

    hiprand_init(seed, i, 0, &state);

    float R_ = 0;

    R_ = cuDFNsys::RandomPowerlaw(minR, maxR, alpha, hiprand_uniform(&state));

    verts[i].Radius = R_;
    //printf("%f\n", verts[i].Radius);

    verts[i].NumVertsTruncated = 4;
    //printf("conductivity_powerlaw_exponent: %f, kappa: %f\n", conductivity_powerlaw_exponent, kappa);

    verts[i].Conductivity = 1.0;

    verts[i].Center.x = cuDFNsys::RandomUniform(-model_L * 0.5, model_L * 0.5,
                                                hiprand_uniform(&state));
    verts[i].Center.y = 0;
    verts[i].Center.z = cuDFNsys::RandomUniform(-model_L * 0.5, model_L * 0.5,
                                                hiprand_uniform(&state));

    verts[i].NormalVec = make_float3(cuDFNsys::RandomUniform(-1.0, 1.0, hiprand_uniform(&state)),
                                     0,
                                     cuDFNsys::RandomUniform(-1.0, 1.0, hiprand_uniform(&state)));
    float norm_f = sqrt(verts[i].NormalVec.x * verts[i].NormalVec.x +
                        verts[i].NormalVec.y * verts[i].NormalVec.y +
                        verts[i].NormalVec.z * verts[i].NormalVec.z);
    verts[i].NormalVec.x /= norm_f;
    verts[i].NormalVec.y /= norm_f;
    verts[i].NormalVec.z /= norm_f;

    //--------------
    float3 rotate_axis = make_float3(-verts[i].NormalVec.y,
                                     verts[i].NormalVec.x,
                                     0);
    float rotate_axis_norm = sqrt(rotate_axis.x * rotate_axis.x +
                                  rotate_axis.y * rotate_axis.y +
                                  rotate_axis.z * rotate_axis.z);
    rotate_axis.x /= rotate_axis_norm;
    rotate_axis.y /= rotate_axis_norm;
    rotate_axis.z /= rotate_axis_norm;

    cuDFNsys::Quaternion qua;
    qua = qua.DescribeRotation(verts[i].NormalVec, 45.0 / 180.0 * M_PI);
    verts[i].Verts3D[0] = qua.Rotate(rotate_axis);

    //--------------
    float norm_vert1 = sqrt(verts[i].Verts3D[0].x * verts[i].Verts3D[0].x +
                            verts[i].Verts3D[0].y * verts[i].Verts3D[0].y +
                            verts[i].Verts3D[0].z * verts[i].Verts3D[0].z);
    norm_vert1 = R_ / norm_vert1;
    verts[i].Verts3D[0].x *= norm_vert1;
    verts[i].Verts3D[0].y *= norm_vert1;
    verts[i].Verts3D[0].z *= norm_vert1;
    verts[i].Verts3D[2].x = -1.0 * verts[i].Verts3D[0].x;
    verts[i].Verts3D[2].y = -1.0 * verts[i].Verts3D[0].y;
    verts[i].Verts3D[2].z = -1.0 * verts[i].Verts3D[0].z;

    verts[i].Verts3D[1] = cuDFNsys::CrossProductFloat3(verts[i].NormalVec,
                                                       verts[i].Verts3D[0]);
    norm_vert1 = sqrt(verts[i].Verts3D[1].x * verts[i].Verts3D[1].x +
                      verts[i].Verts3D[1].y * verts[i].Verts3D[1].y +
                      verts[i].Verts3D[1].z * verts[i].Verts3D[1].z);
    norm_vert1 = R_ / norm_vert1;
    verts[i].Verts3D[1].x *= norm_vert1;
    verts[i].Verts3D[1].y *= norm_vert1;
    verts[i].Verts3D[1].z *= norm_vert1;
    verts[i].Verts3D[3].x = -1.0 * verts[i].Verts3D[1].x;
    verts[i].Verts3D[3].y = -1.0 * verts[i].Verts3D[1].y;
    verts[i].Verts3D[3].z = -1.0 * verts[i].Verts3D[1].z;
    //-----------------------------------------
    for (int j = 0; j < 4; ++j)
    {
        verts[i].Verts3D[j].x += verts[i].Center.x;
        verts[i].Verts3D[j].y += verts[i].Center.y;
        verts[i].Verts3D[j].z += verts[i].Center.z;

        verts[i].Verts3D[j].y = (verts[i].Verts3D[j].y /
                                 abs(verts[i].Verts3D[j].y)) *
                                0.5 * model_L;
        verts[i].Verts3DTruncated[j].x = verts[i].Verts3D[j].x;
        verts[i].Verts3DTruncated[j].y = verts[i].Verts3D[j].y;
        verts[i].Verts3DTruncated[j].z = verts[i].Verts3D[j].z;
    };

    bool gh = cuDFNsys::TruncateFracture(&verts[i], model_L, 0, 1);
    verts[i].ConnectModelSurf[0] = gh;

    gh = cuDFNsys::TruncateFracture(&verts[i], model_L, 0, -1);
    verts[i].ConnectModelSurf[1] = gh;

    gh = cuDFNsys::TruncateFracture(&verts[i], model_L, 1, 1);
    verts[i].ConnectModelSurf[2] = gh;

    gh = cuDFNsys::TruncateFracture(&verts[i], model_L, 1, -1);
    verts[i].ConnectModelSurf[3] = gh;

    gh = cuDFNsys::TruncateFracture(&verts[i], model_L, 2, 1);
    verts[i].ConnectModelSurf[4] = gh;

    gh = cuDFNsys::TruncateFracture(&verts[i], model_L, 2, -1);
    verts[i].ConnectModelSurf[5] = gh;
}; // Fractures2DLike