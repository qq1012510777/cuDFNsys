#include "hip/hip_runtime.h"
/****************************************************************************
* cuDFNsys - simulating flow and transport in 3D fracture networks          *
* Copyright (C) 2022, Tingchang YIN, Sergio GALINDO-TORRES                  *
*                                                                           *
* This program is free software: you can redistribute it and/or modify      *
* it under the terms of the GNU Affero General Public License as            *
* published by the Free Software Foundation, either version 3 of the        *
* License, or (at your option) any later version.                           *
*                                                                           *
* This program is distributed in the hope that it will be useful,           *
* but WITHOUT ANY WARRANTY; without even the implied warranty of            *
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the             *
* GNU Affero General Public License for more details.                       *
*                                                                           *
* You should have received a copy of the GNU Affero General Public License  *
* along with this program.  If not, see <https://www.gnu.org/licenses/>.    *
*****************************************************************************/

#include "Fractures/IdentifyFracPairSphericalDetection.cuh"

// ====================================================
// NAME:        IdentifyFracPairSphericalDetection
// DESCRIPTION: Identify fracture pair where the two circumscribed
//              spheres intersect
// AUTHOR:      Tingchang YIN
// DATE:        09/04/2022
// ====================================================
template <typename T>
__global__ void cuDFNsys::IdentifyFracPairSphericalDetection(cuDFNsys::Fracture<T> *verts,
                                                             int3 *Frac_pairs,
                                                             int InitialPairNO,
                                                             int count)
{
    int idx_TT = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx_TT > count - 1)
        return;

    int idx = idx_TT + InitialPairNO;

    int x_ = floor((pow(2 * (idx + 1), 0.5) + 1 / 2.0));
    int y_ = idx - 0.5 * x_ * (x_ - 1);
    //printf("%d: x_ = %d, y_ =  %d\n",idx, x_, y_);
    Frac_pairs[idx_TT].x = x_;
    Frac_pairs[idx_TT].y = y_;
    Frac_pairs[idx_TT].z = 1;

    cuDFNsys::Vector3<T> dist_two_frac = cuDFNsys::MakeVector3<T>(verts[x_].Center.x - verts[y_].Center.x,
                                                                  verts[x_].Center.y - verts[y_].Center.y,
                                                                  verts[x_].Center.z - verts[y_].Center.z);

    T ddis = pow(dist_two_frac.x * dist_two_frac.x +
                     dist_two_frac.y * dist_two_frac.y +
                     dist_two_frac.z * dist_two_frac.z,
                 0.5);

    if (ddis > (verts[x_].Radius + verts[y_].Radius))
        Frac_pairs[idx_TT].z = 0;
}; // IdentifyFracPairSphericalDetection
template __global__ void cuDFNsys::IdentifyFracPairSphericalDetection<double>(cuDFNsys::Fracture<double> *verts,
                                                                              int3 *Frac_pairs,
                                                                              int InitialPairNO,
                                                                              int count);
template __global__ void cuDFNsys::IdentifyFracPairSphericalDetection<float>(cuDFNsys::Fracture<float> *verts,
                                                                             int3 *Frac_pairs,
                                                                             int InitialPairNO,
                                                                             int count);