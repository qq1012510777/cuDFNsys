#include "hip/hip_runtime.h"
/****************************************************************************
* cuDFNsys - simulating flow and transport in 3D fracture networks          *
* Copyright (C) 2022, Tingchang YIN, Sergio GALINDO-TORRES                  *
*                                                                           *
* This program is free software: you can redistribute it and/or modify      *
* it under the terms of the GNU Affero General Public License as            *
* published by the Free Software Foundation, either version 3 of the        *
* License, or (at your option) any later version.                           *
*                                                                           *
* This program is distributed in the hope that it will be useful,           *
* but WITHOUT ANY WARRANTY; without even the implied warranty of            *
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the             *
* GNU Affero General Public License for more details.                       *
*                                                                           *
* You should have received a copy of the GNU Affero General Public License  *
* along with this program.  If not, see <https://www.gnu.org/licenses/>.    *
*****************************************************************************/

#include "Fractures/Fracture.cuh"

// ====================================================
// NAME:        Theta
// DESCRIPTION: get theta value (radian).
// AUTHOR:      Tingchang YIN
// DATE:        02/04/2022
// ====================================================
template <typename T>
__device__ __host__ cuDFNsys::Vector1<T> cuDFNsys::Fracture<T>::Theta()
{
    return acos(NormalVec.z);
}; // Theta
template __device__ __host__ cuDFNsys::Vector1<double> cuDFNsys::Fracture<double>::Theta();
template __device__ __host__ cuDFNsys::Vector1<float> cuDFNsys::Fracture<float>::Theta();

// ====================================================
// NAME:        Phi
// DESCRIPTION: get Phi Phi (radian).
// AUTHOR:      Tingchang YIN
// DATE:        02/04/2022
// ====================================================
template <typename T>
__device__ __host__ cuDFNsys::Vector1<T> cuDFNsys::Fracture<T>::Phi()
{
    cuDFNsys::Vector1<T> phi = atan2(NormalVec.y, NormalVec.x);
    return (phi > 0 ? phi : phi + 2.0 * M_PI);
}; // Phi
template __device__ __host__ cuDFNsys::Vector1<double> cuDFNsys::Fracture<double>::Phi();
template __device__ __host__ cuDFNsys::Vector1<float> cuDFNsys::Fracture<float>::Phi();

// ====================================================
// NAME:        RoationMatrix
// DESCRIPTION: get RoationMatrix from 3(2) to 2(3).
//              mode is 32 or 23
// AUTHOR:      Tingchang YIN
// DATE:        02/04/2022
// ====================================================
template <typename T>
__device__ __host__ void cuDFNsys::Fracture<T>::RoationMatrix(cuDFNsys::Vector1<T> tmp_R_1[3][3], const int mode)
{
    cuDFNsys::Vector3<T> rotate_axis;
    rotate_axis.x = -NormalVec.y, rotate_axis.y = NormalVec.x, rotate_axis.z = 0.00;

    cuDFNsys::Vector1<T> norm_axis = sqrt(rotate_axis.x * rotate_axis.x + rotate_axis.y * rotate_axis.y);
    //printf("rotate_axis: %.40f,  %.40f,  %.40f\n", rotate_axis.x, rotate_axis.y, rotate_axis.z);
    rotate_axis.x /= norm_axis;
    rotate_axis.y /= norm_axis;

    if (NormalVec.x == 0 && NormalVec.y == 0 && NormalVec.z == 1)
        rotate_axis.x = 0, rotate_axis.y = 0;

    int sign_ = 0;
    if (mode == 32)
        sign_ = -1;
    else if (mode == 23)
        sign_ = 1;

    cuDFNsys::Quaternion<T> Qua;
    Qua = Qua.DescribeRotation(rotate_axis, sign_ * acos(NormalVec.z));
    //printf("sign_ * acos(NormalVec.z): %.40f\n", sign_ * acos(NormalVec.z));

    cuDFNsys::Vector4<T> quater_ = Qua.GetQuaternionNum();
    cuDFNsys::Vector1<T> w = quater_.x,
                         x = quater_.y,
                         y = quater_.z,
                         z = quater_.w;
    //printf("rotate_axis: %.40f,  %.40f,  %.40f\n", rotate_axis.x, rotate_axis.y, rotate_axis.z);

    cuDFNsys::Vector1<T> tmp_R[3][3] = {1 - 2 * y * y - 2 * z * z, 2 * x * y - 2 * w * z, 2 * x * z + 2 * w * y,
                                        2 * x * y + 2 * w * z, 1 - 2 * x * x - 2 * z * z, 2 * y * z - 2 * w * x,
                                        2 * x * z - 2 * w * y, 2 * y * z + 2 * w * x, 1 - 2 * x * x - 2 * y * y};
    for (int i = 0; i < 3; ++i)
        for (int j = 0; j < 3; ++j)
            tmp_R_1[i][j] = tmp_R[i][j];
}; // RoationMatrix
template __device__ __host__ void cuDFNsys::Fracture<double>::RoationMatrix(cuDFNsys::Vector1<double> tmp_R_1[3][3], const int mode);
template __device__ __host__ void cuDFNsys::Fracture<float>::RoationMatrix(cuDFNsys::Vector1<float> tmp_R_1[3][3], const int mode);

// ====================================================
// NAME:        Generate2DVerts
// DESCRIPTION: generate 2D verts.
// AUTHOR:      Tingchang YIN
// DATE:        02/04/2022
// ====================================================
template <typename T>
__device__ __host__ void cuDFNsys::Fracture<T>::Generate2DVerts(cuDFNsys::Vector2<T> *verts2DDD, uint NUM_verts, bool IfTrimed)
{
    cuDFNsys::Vector3<T> rotate_axis;
    rotate_axis.x = -NormalVec.y, rotate_axis.y = NormalVec.x, rotate_axis.z = 0.00;

    cuDFNsys::Vector1<T> norm_axis = sqrt(rotate_axis.x * rotate_axis.x + rotate_axis.y * rotate_axis.y);
    rotate_axis.x /= norm_axis;
    rotate_axis.y /= norm_axis;

    if (NormalVec.x == 0 && NormalVec.y == 0 && NormalVec.z == 1)
        rotate_axis.x = 0, rotate_axis.y = 0;

    cuDFNsys::Quaternion<T> Qua;
    Qua = Qua.DescribeRotation(rotate_axis, -1.0 * acos(this->NormalVec.z));

    if (IfTrimed == false)
        NUM_verts = 4;
    for (uint i = 0; i < NUM_verts; ++i)
    {
        cuDFNsys::Vector3<T> Vertex__;

        if (IfTrimed == false)
        {
            Vertex__.x = this->Verts3D[i].x - this->Center.x,
            Vertex__.y = this->Verts3D[i].y - this->Center.y,
            Vertex__.z = this->Verts3D[i].z - this->Center.z;
        }
        else
        {
            Vertex__.x = this->Verts3DTruncated[i].x - this->Center.x,
            Vertex__.y = this->Verts3DTruncated[i].y - this->Center.y,
            Vertex__.z = this->Verts3DTruncated[i].z - this->Center.z;
        }

        Vertex__ = Qua.Rotate(Vertex__);
        verts2DDD[i].x = Vertex__.x;
        verts2DDD[i].y = Vertex__.y;
    };
}; // Generate2DVerts
template __device__ __host__ void cuDFNsys::Fracture<double>::Generate2DVerts(cuDFNsys::Vector2<double> *verts2DDD, uint NUM_verts, bool IfTrimed);
template __device__ __host__ void cuDFNsys::Fracture<float>::Generate2DVerts(cuDFNsys::Vector2<float> *verts2DDD, uint NUM_verts, bool IfTrimed);