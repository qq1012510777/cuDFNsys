#include "hip/hip_runtime.h"
#include "MHFEM/ReconstructVelocityGrid.cuh"

// ====================================================
// NAME:        ReconstructVelocityGrid
// DESCRIPTION: Reconstruct the velocity field in a grid
// AUTHOR:      Tingchang YIN
// DATE:        20/04/2022
// ====================================================
template <typename T>
__host__ __device__ cuDFNsys::Vector2<T> cuDFNsys::ReconstructVelocityGrid(cuDFNsys::Vector2<T> Point_,
                                                                           cuDFNsys::Vector2<T> Vertex[3],
                                                                           cuDFNsys::Vector3<T> VelocityEdgeNormal)
{
    cuDFNsys::Vector2<T> velocity_;

    cuDFNsys::Vector2<T> A = cuDFNsys::MakeVector2(Vertex[2].x - Vertex[1].x, Vertex[2].y - Vertex[1].y);
    cuDFNsys::Vector2<T> B = cuDFNsys::MakeVector2(Vertex[0].x - Vertex[2].x, Vertex[0].y - Vertex[2].y);
    cuDFNsys::Vector2<T> C = cuDFNsys::MakeVector2(Vertex[1].x - Vertex[0].x, Vertex[1].y - Vertex[0].y);

    cuDFNsys::Vector3<T> edge_length = cuDFNsys::MakeVector3(sqrt(A.x * A.x + A.y * A.y),
                                                             sqrt(B.x * B.x + B.y * B.y),
                                                             sqrt(C.x * C.x + C.y * C.y));
    T Area = cuDFNsys::Triangle2DArea<T>(Vertex[0], Vertex[1], Vertex[2]);

    cuDFNsys::Vector2<T> Phi_1 = cuDFNsys::MakeVector2(edge_length.x / (2 * Area) * (Point_.x - Vertex[0].x) * VelocityEdgeNormal.y,
                                                       edge_length.x / (2 * Area) * (Point_.y - Vertex[0].y) * VelocityEdgeNormal.y);

    cuDFNsys::Vector2<T> Phi_2 = cuDFNsys::MakeVector2(edge_length.y / (2 * Area) * (Point_.x - Vertex[1].x) * VelocityEdgeNormal.z,
                                                       edge_length.y / (2 * Area) * (Point_.y - Vertex[1].y) * VelocityEdgeNormal.z);

    cuDFNsys::Vector2<T> Phi_3 = cuDFNsys::MakeVector2(edge_length.z / (2 * Area) * (Point_.x - Vertex[2].x) * VelocityEdgeNormal.x,
                                                       edge_length.z / (2 * Area) * (Point_.y - Vertex[2].y) * VelocityEdgeNormal.x);

    velocity_.x = Phi_1.x + Phi_2.x + Phi_3.x;
    velocity_.y = Phi_1.y + Phi_2.y + Phi_3.y;
    return velocity_;
}; // ReconstructVelocityGrid
template __host__ __device__ cuDFNsys::Vector2<double> cuDFNsys::ReconstructVelocityGrid<double>(cuDFNsys::Vector2<double> Point_,
                                                                                                 cuDFNsys::Vector2<double> Vertex[3],
                                                                                                 cuDFNsys::Vector3<double> VelocityEdgeNormal);
template __host__ __device__ cuDFNsys::Vector2<float> cuDFNsys::ReconstructVelocityGrid<float>(cuDFNsys::Vector2<float> Point_,
                                                                                               cuDFNsys::Vector2<float> Vertex[3],
                                                                                               cuDFNsys::Vector3<float> VelocityEdgeNormal);