/****************************************************************************
* cuDFNsys - simulating flow and transport in 3D fracture networks          *
* Copyright (C) 2022, Tingchang YIN, Sergio GALINDO-TORRES                  *
*                                                                           *
* This program is free software: you can redistribute it and/or modify      *
* it under the terms of the GNU Affero General Public License as            *
* published by the Free Software Foundation, either version 3 of the        *
* License, or (at your option) any later version.                           *
*                                                                           *
* This program is distributed in the hope that it will be useful,           *
* but WITHOUT ANY WARRANTY; without even the implied warranty of            *
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the             *
* GNU Affero General Public License for more details.                       *
*                                                                           *
* You should have received a copy of the GNU Affero General Public License  *
* along with this program.  If not, see <https://www.gnu.org/licenses/>.    *
*****************************************************************************/

// ====================================================
// NAME:        A quickstart example
// DESCRIPTION: Call cuDFNsys functions to do simulation.
// AUTHOR:      Tingchang YIN
// DATE:        03/11/2022
// ====================================================

#include "cuDFNsys.cuh"
#include <unistd.h>

#ifdef USE_DOUBLES
typedef double _DataType_;
#else
typedef float _DataType_;
#endif
// _DataType_ here is actually double

int main(int argc, char *argv[])
{

    try
    {
        int dev = 0;                     // No. 0 GPU card
        GPUErrCheck(hipSetDevice(dev)); // try to use the first GPU card and check its availability

        int DSIZE = 150;
        // the number of fractures is 150

        _DataType_ L = 30;
        // the domain size is 30 m, the center of the domain is (0, 0, 0)

        _DataType_ minGrid = 2;
        _DataType_ maxGrid = 3;
        // we set the minimum grid size of mesh is 2m, maxmum is 3 m.
        // they are just input parameter, hence the actual grid size may differ from them more or less

        _DataType_ kappa_para = 0;
        // here we just set fisher constant = 0
        // i.e. uniform orientation

        _DataType_ beta = 0.25;
        // the formula is: b = gamma * r ^ beta, b is aperture
        // r is fracture size
        // gamma here is 5e-4, a default value in the function cuDFNsys::Fractures

        int ModeSizeDistri = 0;
        // ModeSizeDistri here means that the distribution of fracture sizes is a power law
        // fracture size here is the radius of fractures

        cuDFNsys::Vector4<_DataType_> ParaSizeDistri =
            cuDFNsys::MakeVector4(1.5,
                                  1.,
                                  15.,
                                  0.);
        // ParaSizeDistri defines the parameter of distribution of fracture sizes
        // 1.5 = the exponent of a power law
        // 1. = the minimum fracture size
        // 15. = the maximum

        int perco_dir = 2;
        // the pre-defined percolation direction is along z axis
        // or say, flow direction is along z axis

        thrust::host_vector<cuDFNsys::Fracture<_DataType_>> Frac_verts_host(DSIZE);
        thrust::device_vector<cuDFNsys::Fracture<_DataType_>> Frac_verts_device(DSIZE);
        cuDFNsys::Fracture<_DataType_> *Frac_verts_device_ptr;
        Frac_verts_device_ptr = thrust::raw_pointer_cast(Frac_verts_device.data());
        // create host and device vectors of cuDFNsys::Fracture
        // here we use double precision, namely, _DataType_
        // a device pointer, 'Frac_verts_device_ptr', pointing to the elements in device vector

        cuDFNsys::Warmup<<<DSIZE / 256 + 1, 256>>>();
        hipDeviceSynchronize();
        // let us warmup the GPU firstly
        // hipDeviceSynchronize() is very important, means that,
        // after the above kernel function is finished, then the following scripts are implemented.

        time_t t;
        time(&t);
        // t is for random seed

        double istart = cuDFNsys::CPUSecond(); // we can count time

        cuDFNsys::Fractures<_DataType_><<<DSIZE / 256 + 1, 256>>>(Frac_verts_device_ptr, // the pointer to device vector
                                                                  (unsigned long)t,      // seed
                                                                  DSIZE,                 // number of fracture
                                                                  L,                     // domain size
                                                                  ModeSizeDistri,        // distribution pattern of fracture sizes
                                                                  ParaSizeDistri,        // parameters of distribution of fracture sizes
                                                                  kappa_para,            // kappa value of fisher distribution
                                                                  beta);                 // beta

        hipDeviceSynchronize();             // now we finished generating fractures
        Frac_verts_host = Frac_verts_device; // copy data from device to host

        double ielaps = cuDFNsys::CPUSecond() - istart;
        cout << "Running time of fracture generation: " << ielaps << " sec\n";

        std::map<pair<size_t, size_t>, pair<cuDFNsys::Vector3<_DataType_>, cuDFNsys::Vector3<_DataType_>>> Intersection_map;
        // a map for storing intersections,
        // Intersection_map.first = a pair of fracture No.
        // Intersection_map.second = a pair of coordinates, i.e. the ends of the intersection

        istart = cuDFNsys::CPUSecond();
        cuDFNsys::IdentifyIntersection<_DataType_> identifyInters{Frac_verts_host.size(), // number of fractures
                                                                  Frac_verts_device_ptr,  // pointer of device vector of fractures
                                                                  false,                  // if you want to use truncated fractures? here is false,
                                                                  Intersection_map};
        // this process is based on GPU
        ielaps = cuDFNsys::CPUSecond() - istart;
        cout << "Running time of identification of intersections: " << ielaps << " sec\n";

        std::vector<std::vector<size_t>> ListClusters; // will store all fracture clusters
        std::vector<size_t> Percolation_cluster;       // will store the No. of percolating cluster

        istart = cuDFNsys::CPUSecond();
        cuDFNsys::Graph<_DataType_> G{(size_t)DSIZE, Intersection_map};
        G.UseDFS(ListClusters);
        // DFS algorithm to identify clusters

        cuDFNsys::IdentifyPercolationCluster<_DataType_> IdentiClu{ListClusters,         // all clusters
                                                                   Frac_verts_host,      // host vector of fractures
                                                                   perco_dir,            // percolation direction / flow direction
                                                                   Percolation_cluster}; // percolation cluster
        ielaps = cuDFNsys::CPUSecond() - istart;
        cout << "Running time of identification of clusters: " << ielaps << " sec\n";

        // now a DFN is randomly generated, let's see it
        cuDFNsys::MatlabPlotDFN<_DataType_> As{"DFN_I.h5",          // the .h5 file, with suffix
                                               "DFN_I.m",           // matlab script to visualize the DFN, with suffix
                                               Frac_verts_host,     // host vector of fractures
                                               Intersection_map,    // intersection map
                                               ListClusters,        // clusters
                                               Percolation_cluster, // No. or say, ID, of percolating cluster
                                               false,               // if show truncated fractures?
                                               true,                // if show intersections?
                                               true,                // if show clusters?
                                               true,                // if show orientations data?
                                               L,                   // domain size
                                               perco_dir,           // flow direction
                                               true,                // true means I also want to see DFN with python script, a .py file will be generated
                                               "DFN_I"};            // the name of python script, without suffix

        // then I want to identify intersections of TRUNCATED fractures!!!
        // as well as the percolation cluster
        // this is important for flow simulation!!!

        Intersection_map.clear();
        ListClusters.clear();
        Percolation_cluster.clear();
        // for the sake of safety, clear them first

        /// now let's consider truncated fractures!!!
        istart = cuDFNsys::CPUSecond();
        cuDFNsys::IdentifyIntersection<_DataType_> identifyInters2___{Frac_verts_host.size(),
                                                                      Frac_verts_device_ptr,
                                                                      true,
                                                                      Intersection_map};
        ielaps = cuDFNsys::CPUSecond() - istart;
        cout << "Running time of identification of intersections with truncated fractures: " << ielaps << " sec\n";

        istart = cuDFNsys::CPUSecond();
        cuDFNsys::Graph<_DataType_> G2__{(size_t)DSIZE, Intersection_map};
        G2__.UseDFS(ListClusters);
        // DFS algorithm to identify clusters

        cuDFNsys::IdentifyPercolationCluster<_DataType_> IdentiClu2____{ListClusters,
                                                                        Frac_verts_host,
                                                                        perco_dir,
                                                                        Percolation_cluster};
        ielaps = cuDFNsys::CPUSecond() - istart;
        cout << "Running time of identification of clusters with truncated fractures: " << ielaps << " sec\n";

        cuDFNsys::MatlabPlotDFN<_DataType_> As2__{"DFN_II.h5",         // the .h5 file, with suffix
                                                  "DFN_II.m",          // matlab script to visualize the DFN, with suffix
                                                  Frac_verts_host,     // host vector of fractures
                                                  Intersection_map,    // intersection map
                                                  ListClusters,        // clusters
                                                  Percolation_cluster, // No. or say, ID, of percolating cluster
                                                  true,                // if show truncated fractures?
                                                  true,                // if show intersections?
                                                  true,                // if show clusters?
                                                  true,                // if show orientations data?
                                                  L,                   // domain size
                                                  perco_dir,           // flow direction
                                                  true,                // true means I also want to see DFN with python script, a .py file will be generated
                                                  "DFN_II"};           // the name of python script, without suffix

        Frac_verts_device.clear();
        Frac_verts_device.shrink_to_fit();
        // device vector of fractures now is not neccessary

        if (Percolation_cluster.size() > 0) // please be sure that there is at least one spanning cluster
        {
            std::vector<size_t> Fracs_percol; // will store ID / No of fractures in percolating cluster

            istart = cuDFNsys::CPUSecond();
            cuDFNsys::GetAllPercolatingFractures GetPer{Percolation_cluster,
                                                        ListClusters,
                                                        Fracs_percol};
            // this function is simple, just collecting fractures in the percolating clusters

            std::vector<pair<int, int>> IntersectionPair_percol;
            // will store the intersection pair of fractures in percolation clusters

            cuDFNsys::RemoveDeadEndFrac<_DataType_> RDEF{Fracs_percol,            // fractures' ID in percolating cluster
                                                         IntersectionPair_percol, // intersection pair
                                                         (size_t)perco_dir,       // flow direction
                                                         Frac_verts_host,         // host vector of fractures
                                                         Intersection_map};       // map of intersection
            // the above function removes dead end fractures

            ielaps = cuDFNsys::CPUSecond() - istart;
            cout << "Running time of removing dead-end fractures: " << ielaps << " sec\n";

            istart = cuDFNsys::CPUSecond();
            cuDFNsys::Mesh<_DataType_> mesh{Frac_verts_host,         // host vector of fractures, after removing fractures of dead end
                                            IntersectionPair_percol, // intersection pair
                                            &Fracs_percol,           // fractures' ID in percolating cluster
                                            minGrid,                 // minimum grid size
                                            maxGrid,                 // maximum grid size
                                            perco_dir,               // flow direction
                                            L};                      // domain size
            // mesh finished
            mesh.MatlabPlot("DFN_mesh.h5",   // h5 file
                            "DFN_mesh.m",    // name of matlab script
                            Frac_verts_host, // fracture vector on the host side
                            L,               // domain size
                            true,            // if check 2D coordinates, because 3D fractures can be mapped to 2D plane
                            true,            // if check the edge attributes? Neumann, Dirichlet?
                            true,            // if I want to see mesh with Python?
                            "DFN_mesh");     // name of python script without suffix
            ielaps = cuDFNsys::CPUSecond() - istart;
            cout << "Running time of mesh: " << ielaps << " sec\n";

            istart = cuDFNsys::CPUSecond();
            cuDFNsys::MHFEM<_DataType_> fem{mesh,            // mesh object
                                            Frac_verts_host, // fractures
                                            100,             // hydraulic head of inlet = 100 m
                                            20,              // hydraulic head of outlet = 20 m
                                            perco_dir,       // flow direction
                                            L};              // domain size

            fem.MatlabPlot("MHFEM.h5",      // h5 file
                           "MHFEM.m",       // matlab script to see mhfem result
                           Frac_verts_host, // fractures
                           mesh,            // mesh object
                           L,               // domain size
                           true,            // if use python to do visualization
                           "MHFEM");        // name of python script, without suffix

            ielaps = cuDFNsys::CPUSecond() - istart;
            cout << "Running time of mhfem: " << ielaps << " sec\n";

            cuDFNsys::OutputObjectData<_DataType_> lk;
            lk.OutputFractures("FracturesForParticle.h5", Frac_verts_host, L);
            // the above two command just in order to output fractures information to transform 2D particle data to 3D

            istart = cuDFNsys::CPUSecond();
            cuDFNsys::ParticleTransport<_DataType_> p{(unsigned long)t,          // random seed
                                                      atoi(argv[1]),             // number of particle
                                                      atoi(argv[2]),             // number of time steps
                                                      (_DataType_)atof(argv[3]), // delta T
                                                      (_DataType_)atof(argv[4]), // molecular diffusion
                                                      Frac_verts_host,           // fractures
                                                      mesh,                      // mesh object
                                                      fem,                       // mhfem object
                                                      (uint)perco_dir,           // flow direction
                                                      -0.5 * L,                  // the target plane, z = -0.5 * L
                                                      "Particle_tracking",       // use particle tracking algorithm
                                                      "Flux-weighted"};          // the injection mode is flux-weighted

            p.MatlabPlot("MHFEM.h5",            // h5 file of mhfem
                         "ParticlesMovement.m", // matlab script
                         mesh,                  // mesh result
                         fem,                   // mhfem object
                         L,                     // domain size
                         true,
                         "ParticlesMovement");

            // note that right now data in the output file () is not 3D
            // you have to transform 2D data to 3D
            // which can be done by run the executable file 'Transform2DH5ParticleDataTo3D'
            // then, the ParticlesMovement.m can be implemented in matlab
            // or ParticlesMovement.py can be run
            // The 'compileCode.sh' will compile the 'Transform2DH5ParticleDataTo3D' code and run it after data (particle positions) output

            ielaps = cuDFNsys::CPUSecond() - istart;
            cout << "Running time of particle tracking: " << ielaps << " sec\n";
        }
    }
    catch (cuDFNsys::ExceptionsIgnore &e)
    {
        cout << e.what() << endl;
    }
    catch (cuDFNsys::ExceptionsPause &e)
    {
        cout << e.what() << endl;
    }
    catch (...)
    {
        throw;
    };
    return 0;
};