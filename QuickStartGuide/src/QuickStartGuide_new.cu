#include "hip/hip_runtime.h"
/****************************************************************************
 * cuDFNsys - simulating flow and transport in 3D fracture networks          *
 * Copyright (C) 2022, Tingchang YIN, Sergio GALINDO-TORRES                  *
 *                                                                           *
 * This program is free software: you can redistribute it and/or modify      *
 * it under the terms of the GNU Affero General Public License as            *
 * published by the Free Software Foundation, either version 3 of the        *
 * License, or (at your option) any later version.                           *
 *                                                                           *
 * This program is distributed in the hope that it will be useful,           *
 * but WITHOUT ANY WARRANTY; without even the implied warranty of            *
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the             *
 * GNU Affero General Public License for more details.                       *
 *                                                                           *
 * You should have received a copy of the GNU Affero General Public License  *
 * along with this program.  If not, see <https://www.gnu.org/licenses/>.    *
 *****************************************************************************/

// ====================================================
// NAME:        A quickstart example to generate DFNs
// DESCRIPTION: Call cuDFNsys functions to do simulation.
// AUTHOR:      Tingchang YIN
// DATE:        13/10/2023
// ====================================================

#include "cuDFNsys.cuh"
#include <fstream>
#include <iostream>
#include <limits.h>
#include <unistd.h>

int main(int argc, char *argv[])
{
    time_t t;
    time(&t);

    cuDFNsys::DFN<double> my_dfn;

    my_dfn.NumFractures = {70, 80};
    my_dfn.Kappa = {20, 10};
    my_dfn.MeanOrientationOfFisherDistribution = {make_double3(0., 0., 1.),
                                                  make_double3(1., 0., 0.)};
    my_dfn.DomainSizeX = 30;
    my_dfn.DomainDimensionRatio = make_double3(1., 1., 2.);
    my_dfn.Beta = {0.2, 0.3};
    my_dfn.Gamma = {2.0e-5, 3.0e-6};
    my_dfn.ModeOfSizeDistribution = {0, 1};
    my_dfn.SizeDistributionParameters = {make_double4(1.5, 1., 15., 0.),
                                         make_double4(8.5, 5.5, 1., 15.)};
    my_dfn.PercoDir = 2;
    my_dfn.RandomSeed = (unsigned long)t;
    my_dfn.FractureGeneration();
    my_dfn.IdentifyIntersectionsClusters(true);
    my_dfn.Visualization("DFN_VISUAL", "DFN_VISUAL", "DFN_VISUAL", true, true,
                         true, true);

    cuDFNsys::MeshDFN<double> meshGen;
    meshGen.MinElementSize = 1;
    meshGen.MaxElementSize = 3;
    meshGen.MeshGeneration(my_dfn);
    meshGen.Visualization(my_dfn, "DFN_MESH_VISUAL", "DFN_MESH_VISUAL",
                          "DFN_MESH_VISUAL", true, true);

    cuDFNsys::FlowDFN<double> flowDFN;
    flowDFN.InletHead = 60;
    flowDFN.OutletHead = 0;
    flowDFN.FlowSimulation(my_dfn, meshGen);
    flowDFN.Visualization(my_dfn, meshGen, "DFN_FLOW_VISUAL", "DFN_FLOW_VISUAL",
                          "DFN_FLOW_VISUAL");

    cuDFNsys::PTDFN<double> particleTracking;
    particleTracking.NumParticles = 20000;
    particleTracking.NumTimeSteps = 200;
    particleTracking.PecletNumber = 300;
    particleTracking.LengthScalePe = 30;
    particleTracking.VelocityScalePe = flowDFN.MeanVelocity;
    particleTracking.MolecularDiffusion = particleTracking.LengthScalePe /
                                          particleTracking.PecletNumber *
                                          particleTracking.VelocityScalePe;
    particleTracking.FactorTimeScaleCrossElement = 2;
    particleTracking.TimeScaleCrossElement =
        pow(meshGen.MeanGridSize, 0.5) / flowDFN.MaxVelocity;
    particleTracking.DeltaT = particleTracking.TimeScaleCrossElement /
                              particleTracking.FactorTimeScaleCrossElement;
    particleTracking.FluexWeightedOrUniformInjection = true;
    particleTracking.OutputAllPTInformationOrFPTCurve = true;
    particleTracking.SpacingOfControlPlanes = 30;
    particleTracking.IfOutputVarianceOfDisplacementsEachStep = true;
    particleTracking.IfInjectAtCustomedPlane = true;
    particleTracking.CustomedPlaneInjection = 23;
    particleTracking.IfUseFluxWeightedOrEqualProbableMixingIntersection = true;

    particleTracking.ParticleTracking(my_dfn, meshGen, flowDFN);
    particleTracking.Visualization(my_dfn, meshGen, flowDFN,
                                   "DFN_DISPERSION_VISUAL",
                                   "DFN_DISPERSION_VISUAL", "DFN_FLOW_VISUAL");
    return 0;
};
